#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_shuffledPos;
glm::vec3 *dev_shuffledVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  hipDeviceSynchronize();

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  // 2.3 Additional buffers.
  hipMalloc((void**)&dev_shuffledPos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_shuffledPos failed!");

  hipMalloc((void**)&dev_shuffledVel, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_shuffledVel failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	glm::vec3 boidPos = pos[iSelf];

	glm::vec3 perceivedCenter = glm::vec3(0.0f);
	glm::vec3 c = glm::vec3(0.0f);
	glm::vec3 percVel = glm::vec3(0.0f);;

	int rule1Counter = 0;
	int rule3Counter = 0;
	for (int i = 0; i < N; i++) {
		float distance = glm::distance(pos[i], boidPos);
		// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
		if (i != iSelf && distance < rule1Distance) {
			perceivedCenter += pos[i];
			rule1Counter++;
		}
		// Rule 2
		if (i != iSelf && distance < rule2Distance) {
			c -= (pos[i] - boidPos);
		}
		// Rule 3
		if (i != iSelf && distance < rule3Distance) {
			percVel += vel[i];
			rule3Counter++;
		}
	}
	
	if (rule1Counter != 0) {
		perceivedCenter /= rule1Counter;
		perceivedCenter = perceivedCenter - boidPos;
	}
	if (rule3Counter != 0) {
		percVel /= rule3Counter;
	}
	
 	return perceivedCenter*rule1Scale + c * rule2Scale + percVel * rule3Scale;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

	
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	vel2[index] = vel1[index] + computeVelocityChange(N, index, pos, vel1);
	if (glm::length(vel2[index]) > maxSpeed) {
		vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
	}
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__device__ int getGridCellQuadrant(float gridCellPos, float origin) {
	if (gridCellPos > origin) {
		return 1;
	}
	else {
		return -1;
	}
}


// Added 9/9/17
__device__ void checkGridCellAndUpdateVel(int x, int y, int z, int gridResolution,
	int *gridCellStartIndices, int *gridCellEndIndices,
	int *particleArrayIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2, int iSelf) {
	if (x < 0 || x > gridResolution || y < 0 || y > gridResolution || z < 0 || z > gridResolution) {
		return;
	}
	else {
		int gridIndex = gridIndex3Dto1D(x, y, z, gridResolution);
		int start = gridCellStartIndices[gridIndex];
		int end = gridCellEndIndices[gridIndex];
		if (start == -1 || end == -1) {
			return;
		}

		// TODO(Wenli): Repetive code, consider refactoring.
		glm::vec3 boidPos = pos[iSelf];
		glm::vec3 perceivedCenter = glm::vec3(0.0f);
		glm::vec3 c = glm::vec3(0.0f);
		glm::vec3 percVel = glm::vec3(0.0f);;

		int rule1Counter = 0;
		int rule3Counter = 0;

		for (int j = start; j <= end; j++) {
			// Bug was here -_-
			int i = particleArrayIndices[j];
			float distance = glm::distance(pos[i], boidPos);
			// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
			if (i != iSelf && distance < rule1Distance) {
				perceivedCenter += pos[i];
				rule1Counter++;
			}
			// Rule 2
			if (i != iSelf && distance < rule2Distance) {
				c -= (pos[i] - boidPos);
			}
			// Rule 3
			if (i != iSelf && distance < rule3Distance) {
				percVel += vel1[i];
				rule3Counter++;
			}
		}
		if (rule1Counter != 0) {
			perceivedCenter /= rule1Counter;
			perceivedCenter = perceivedCenter - boidPos;
		}
		if (rule3Counter != 0) {
			percVel /= rule3Counter;
		}
		vel2[iSelf] += perceivedCenter*rule1Scale + c * rule2Scale + percVel * rule3Scale;
	}
}

__device__ void checkGridCellAndUpdateVelCoherent(int x, int y, int z, int gridResolution,
	int *gridCellStartIndices, int *gridCellEndIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2, int iSelf) {
	if (x < 0 || x > gridResolution || y < 0 || y > gridResolution || z < 0 || z > gridResolution) {
		return;
	}
	else {
		int gridIndex = gridIndex3Dto1D(x, y, z, gridResolution);
		int start = gridCellStartIndices[gridIndex];
		int end = gridCellEndIndices[gridIndex];
		if (start == -1 || end == -1) {
			return;
		}

		// TODO(Wenli): Repetive code, consider refactoring.
		glm::vec3 boidPos = pos[iSelf];
		glm::vec3 perceivedCenter = glm::vec3(0.0f);
		glm::vec3 c = glm::vec3(0.0f);
		glm::vec3 percVel = glm::vec3(0.0f);;

		int rule1Counter = 0;
		int rule3Counter = 0;

		for (int i = start; i <= end; i++) {
			float distance = glm::distance(pos[i], boidPos);
			// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
			if (i != iSelf && distance < rule1Distance) {
				perceivedCenter += pos[i];
				rule1Counter++;
			}
			// Rule 2
			if (i != iSelf && distance < rule2Distance) {
				c -= (pos[i] - boidPos);
			}
			// Rule 3
			if (i != iSelf && distance < rule3Distance) {
				percVel += vel1[i];
				rule3Counter++;
			}
		}
		if (rule1Counter != 0) {
			perceivedCenter /= rule1Counter;
			perceivedCenter = perceivedCenter - boidPos;
		}
		if (rule3Counter != 0) {
			percVel /= rule3Counter;
		}
		vel2[iSelf] += perceivedCenter*rule1Scale + c * rule2Scale + percVel * rule3Scale;
	}
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index > N) {
		return;
	}

	int x = glm::floor((pos[index].x - gridMin.x)*inverseCellWidth);
	int y = glm::floor((pos[index].y - gridMin.y)*inverseCellWidth);
	int z = glm::floor((pos[index].z - gridMin.z)*inverseCellWidth);

	gridIndices[index] = gridIndex3Dto1D(x, y, z, gridResolution);
	indices[index] = index;

}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
// wz: start end buffers, set to -1?
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index > N) {
		return;
	}

	int gridIdx = particleGridIndices[index];
	int prevIdx = particleGridIndices[index - 1];
	
	if (index > 0 && prevIdx != gridIdx) {
		gridCellStartIndices[gridIdx] = index;
		gridCellEndIndices[prevIdx] = index - 1;
	}
	if (index == 0) {
		gridCellStartIndices[gridIdx] = index;
	}

}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.

  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index > N) {
		return;
	}

	// get particle pos
	glm::vec3 partPos = pos[index];

	int x = glm::floor((partPos.x - gridMin.x)*inverseCellWidth);
	int y = glm::floor((partPos.y - gridMin.y)*inverseCellWidth);
	int z = glm::floor((partPos.z - gridMin.z)*inverseCellWidth);

	// identify grid cell
	int gridCell = gridIndex3Dto1D(x, y, z, gridResolution);

	// calculate grid positions of up to eight neighbors

	//glm::vec3 center = glm::vec3(x + 0.5, y + 0.5, z + 0.5);

	// qX, qY, qZ are -1 or 1 depending on what quadrant of the cell the boid is in 
	glm::vec3 floatPos = partPos - gridMin;
	glm::vec3 center = glm::vec3(x*cellWidth + cellWidth / 2, y*cellWidth + cellWidth / 2, z *cellWidth + cellWidth / 2);
	glm::vec3 quadVec = floatPos - center;

	int qX = getGridCellQuadrant(quadVec.x, 0);
	int qY = getGridCellQuadrant(quadVec.y, 0);
	int qZ = getGridCellQuadrant(quadVec.z, 0);

	// Set vel2.
	vel2[index] = vel1[index];

	// check grid cells at x (+ qX), y (+ qY), z (+ qZ)
	// add to Vel2
	checkGridCellAndUpdateVel(x, y, z, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x, y, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x, y + qY, z, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x, y + qY, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x + qX, y, z, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x + qX, y, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x + qX, y + qY, z, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVel(x + qX, y + qY, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1, vel2, index);

	if (glm::length(vel2[index]) > maxSpeed) {
		vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
	}
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index > N) {
		return;
	}

	// get particle pos
	glm::vec3 partPos = pos[index];

	int x = glm::floor((partPos.x - gridMin.x)*inverseCellWidth);
	int y = glm::floor((partPos.y - gridMin.y)*inverseCellWidth);
	int z = glm::floor((partPos.z - gridMin.z)*inverseCellWidth);

	// identify grid cell
	int gridCell = gridIndex3Dto1D(x, y, z, gridResolution);

	// calculate grid positions of up to eight neighbors
	glm::vec3 floatPos = partPos - gridMin;
	glm::vec3 center = glm::vec3(x*cellWidth + cellWidth / 2, y*cellWidth + cellWidth / 2, z *cellWidth + cellWidth / 2);
	glm::vec3 quadVec = floatPos - center;

	// qX, qY, qZ are -1 or 1 depending on what quadrant of the cell the boid is in 
	int qX = getGridCellQuadrant(quadVec.x, 0);
	int qY = getGridCellQuadrant(quadVec.y, 0);
	int qZ = getGridCellQuadrant(quadVec.z, 0);

	// Set vel2.
	vel2[index] = vel1[index];

	// check grid cells at x (+ qX), y (+ qY), z (+ qZ)
	// add to Vel2
	checkGridCellAndUpdateVelCoherent(x, y, z, gridResolution, gridCellStartIndices, gridCellEndIndices,  pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x, y, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x, y + qY, z, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x, y + qY, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x + qX, y, z, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x + qX, y, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x + qX, y + qY, z, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);
	checkGridCellAndUpdateVelCoherent(x + qX, y + qY, z + qZ, gridResolution, gridCellStartIndices, gridCellEndIndices, pos, vel1, vel2, index);

	if (glm::length(vel2[index]) > maxSpeed) {
		vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
	}
}

__global__ void kernShufflePositionAndVelocity(int N, int *particleArrayIndices, glm::vec3 *pos, glm::vec3 *shuffledPos, glm::vec3 *vel1, glm::vec3 *shuffledVel) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) {
		return;
	}

	int shuffledIdx = particleArrayIndices[index];
	shuffledPos[index] = pos[shuffledIdx];
	shuffledVel[index] = vel1[shuffledIdx];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel2);

	glm::vec3 *temp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp;

}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	// Label indices
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > 
		(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);

	// Sort using thrust.
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

	// Set all the dev_gridCellStartIndices and EndIndices to -1
	dim3 cellBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize); 

	kernResetIntBuffer << <cellBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <cellBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

	// Find start and end indices.
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >  
		(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// Velocity updates.
	kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> >
		(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
			dev_gridCellStartIndices, dev_gridCellEndIndices, 
			dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

	// Update position.
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel2);

	// Ping pong.
	glm::vec3 *temp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp;

}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	// Label indices
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> >
		(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);

	// Sort using thrust.
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

	// Set all the dev_gridCellStartIndices and EndIndices to -1
	dim3 cellBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);

	kernResetIntBuffer << <cellBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <cellBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

	// Find start and end indices.
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >
		(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// Rearrange positions/velocity
	kernShufflePositionAndVelocity << <fullBlocksPerGrid, blockSize >> >
		(numObjects, dev_particleArrayIndices, dev_pos, dev_shuffledPos, dev_vel1, dev_shuffledVel);

	// Velocity updates.
	kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> >
		(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
			dev_gridCellStartIndices, dev_gridCellEndIndices,
			dev_shuffledPos, dev_shuffledVel, dev_vel2);

	// Update position with shuffledPos.
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_shuffledPos, dev_vel2);

	// Ping pong.
	glm::vec3 *temp1 = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp1;

	//glm::vec3 *temp2 = dev_vel2;
	//dev_vel2 = dev_shuffledVel;
	//dev_shuffledVel = temp2;

	//dev_vel2 = dev_shuffledVel;

	glm::vec3 *temp = dev_pos;
	dev_pos = dev_shuffledPos;
	dev_shuffledPos = temp;

}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  
  hipFree(dev_shuffledVel);
  hipFree(dev_shuffledPos);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
