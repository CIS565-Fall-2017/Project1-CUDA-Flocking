#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_coherentPos;
glm::vec3 *dev_coherentVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);

  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipMalloc((void**)&dev_gridCellStartIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_coherentPos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_coherentVel, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	// Rule 2: boids try to stay a distance d away from each other
	// Rule 3: boids try to match the speed of surrounding boids
	// Fields that we need to know about the selected boid.
	glm::vec3 currentPosition = pos[iSelf];
	glm::vec3 currentVelocity = vel[iSelf];
	float perceivedMassNeighbors = 0.0f;
	glm::vec3 perceivedCenterTotal = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceivedCenterVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separationVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
	float perceivedVelocityNeighbors = 0.0f;
	glm::vec3 perceivedNeighborVelocityTotal = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceivedNeighborVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

	// Check all of the other boids.
	for (int i = 0; i < N; ++i) {
		if (i != iSelf) {
			glm::vec3 neighborPosition = pos[i];
			glm::vec3 neighborVelocity = vel[i];
			float distanceNeighbor = glm::distance(currentPosition, neighborPosition);

			// Find neighbors which factor into the perceived center of mass.
			if (distanceNeighbor < rule1Distance) {
				++perceivedMassNeighbors;
				perceivedCenterTotal += neighborPosition;
			}

			// Maintain separation.
			if (distanceNeighbor < rule2Distance) {
				separationVelocity -= (neighborPosition - currentPosition);
			}

			// Follow other boids.
			if (distanceNeighbor < rule3Distance) {
				++perceivedVelocityNeighbors;
				perceivedNeighborVelocityTotal += neighborVelocity;
			}
		}
	}

	// Find the average of the perceived mass total for the center, then convert to velocity.
	if (perceivedMassNeighbors > 0) {
		perceivedCenterVelocity = ((perceivedCenterTotal / perceivedMassNeighbors) - currentPosition) * rule1Scale;
	}

	// Scale the separation velocity.
	separationVelocity *= rule2Scale;

	// Find the average of the perceived velocity total.
	if (perceivedVelocityNeighbors > 0) {
		perceivedNeighborVelocity = (perceivedNeighborVelocityTotal / perceivedVelocityNeighbors) * rule3Scale;
	}

	// The current velocity is added because this function doesn't actually compute the velocity change.
	// It's supposed to compute the new velocity instead.
	return (currentVelocity + perceivedCenterVelocity + separationVelocity + perceivedNeighborVelocity);
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
	// Compute a new velocity based on pos and vel1
	// Clamp the speed
	// Record the new velocity into vel2. Question: why NOT vel1?
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 newVelocity = computeVelocityChange(N, index, pos, vel1);
	if (glm::length(newVelocity) > maxSpeed) {
		newVelocity = (maxSpeed * glm::normalize(newVelocity));
	}
	vel2[index] = newVelocity;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	indices[index] = index;

	glm::vec3 currentPosition = pos[index];
	glm::vec3 gridCoordinates = glm::floor((currentPosition - gridMin) * inverseCellWidth);
	int gridIndex = gridIndex3Dto1D(gridCoordinates.x, gridCoordinates.y, gridCoordinates.z, gridResolution);
	gridIndices[index] = gridIndex;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
	// TODO-2.1
	// Identify the start point of each cell in the gridIndices array.
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"

	// Originally I used this code. However, a post on the
	// Google group identified branching like this as a major slowdown.
	/* int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	// Always explicitly specify index 0.
	int gridIndex = particleGridIndices[index];
	if (index == 0) {
		gridCellStartIndices[gridIndex] = index;
	} else {

		// The last index might be part of a series of same numbers; need to explicitly set.
		if (index == (N - 1)) {
			gridCellEndIndices[gridIndex] = index;
		} else {

			// If guaranteed not to be out of bounds, different numbers must set new indicies.
			int previousGridIndex = particleGridIndices[index - 1];
			if (gridIndex != previousGridIndex) {
				gridCellEndIndices[previousGridIndex] = index - 1;
				gridCellStartIndices[gridIndex] = index;
			}
		}
	} */

	// I see significant performance gains from minimizing the branches.
	// I also inlined many of the variables, just in case that mattered.
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	if (index == 0) {
		gridCellStartIndices[particleGridIndices[index]] = 0;
	}
	
	if (index == (N - 1)) {
		gridCellEndIndices[particleGridIndices[index]] = (N - 1);
	} 
	
	if (index < N 
		&& particleGridIndices[index - 1] != particleGridIndices[index]) {
		gridCellStartIndices[particleGridIndices[index]] = index;
		gridCellEndIndices[particleGridIndices[index - 1]] = index - 1;
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	int *particleArrayIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
	// TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.
	// - For each cell, read the start/end indices in the boid pointer array.
	// - Access each boid in the cell and compute velocity change from
	//   the boids rules, if this boid is within the neighborhood distance.
	// - Clamp the speed change before putting the new speed in vel2
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	// Retrieve the current position as it relates to the grid, and the grid cell.
	glm::vec3 currentPosition = pos[index];
	glm::vec3 cellCenter = currentPosition + gridMin + (glm::vec3(0.5f, 0.5f, 0.5f) * cellWidth);
	glm::vec3 gridCoordinates = glm::floor((currentPosition - gridMin) * inverseCellWidth);

	// Figure out which octant of the cell the boid is in; two options along each of x, y, and z.
	// These options determine the neighboring 8 cells to check so that's encoded here.
	glm::ivec3 octant = glm::ivec3(gridCoordinates.x - 1, gridCoordinates.y - 1, gridCoordinates.z - 1);
	if (currentPosition.x > cellCenter.x) {
		++octant.x;
	}
	if (currentPosition.y > cellCenter.y) {
		++octant.y;
	}
	if (currentPosition.z > cellCenter.z) {
		++octant.z;
	}

	// Fields that we need to know about the selected boid.
	glm::vec3 currentVelocity = vel1[index];
	float perceivedMassNeighbors = 0.0f;
	glm::vec3 perceivedCenterTotal = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceivedCenterVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separationVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
	float perceivedVelocityNeighbors = 0.0f;
	glm::vec3 perceivedNeighborVelocityTotal = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceivedNeighborVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

	// Check the neighboring 8 cells
	for (int i = 0; i < 2; ++i) {
		for (int j = 0; j < 2; ++j) {
			for (int k = 0; k < 2; ++k) {

				// Restrict the set of neighbors to stay within the bounds of the grid.
				int neighborIndex = gridIndex3Dto1D(glm::clamp(octant.x + i, 0, gridResolution - 1), glm::clamp(octant.y + j, 0, gridResolution - 1), 
					glm::clamp(octant.z + k, 0, gridResolution - 1), gridResolution);

				// Simulate these neighboring cell's boids.
				for (int boidIndex = gridCellStartIndices[neighborIndex]; boidIndex < gridCellEndIndices[neighborIndex] + 1; ++boidIndex) {
					if (boidIndex != -1 && particleArrayIndices[boidIndex] != index) {
						int nearbyBoidIndex = particleArrayIndices[boidIndex];
						glm::vec3 neighborPosition = pos[nearbyBoidIndex];
						glm::vec3 neighborVelocity = vel1[nearbyBoidIndex];
						float distanceNeighbor = glm::distance(currentPosition, neighborPosition);

						// Find neighbors which factor into the perceived center of mass.
						if (distanceNeighbor < rule1Distance) {
							++perceivedMassNeighbors;
							perceivedCenterTotal += neighborPosition;
						}

						// Maintain separation.
						if (distanceNeighbor < rule2Distance) {
							separationVelocity -= (neighborPosition - currentPosition);
						}

						// Follow other boids.
						if (distanceNeighbor < rule3Distance) {
							++perceivedVelocityNeighbors;
							perceivedNeighborVelocityTotal += neighborVelocity;
						}
					}
				}
			}
		}
	}

	// Find the average of the perceived mass total for the center, then convert to velocity.
	if (perceivedMassNeighbors > 0) {
		perceivedCenterVelocity = ((perceivedCenterTotal / perceivedMassNeighbors) - currentPosition) * rule1Scale;
	}

	// Scale the separation velocity.
	separationVelocity *= rule2Scale;

	// Find the average of the perceived velocity total.
	if (perceivedVelocityNeighbors > 0) {
		perceivedNeighborVelocity = (perceivedNeighborVelocityTotal / perceivedVelocityNeighbors) * rule3Scale;
	}

	// The current velocity is added because this function doesn't actually compute the velocity change.
	// It's supposed to compute the new velocity instead.
	glm::vec3 newVelocity = (currentVelocity + perceivedCenterVelocity + separationVelocity + perceivedNeighborVelocity);
	if (glm::length(newVelocity) > maxSpeed) {
		newVelocity = (maxSpeed * glm::normalize(newVelocity));
	}
	vel2[index] = newVelocity;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
	// TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.
	// - For each cell, read the start/end indices in the boid pointer array.
	//   DIFFERENCE: For best results, consider what order the cells should be
	//   checked in to maximize the memory benefits of reordering the boids data.
	// - Access each boid in the cell and compute velocity change from
	//   the boids rules, if this boid is within the neighborhood distance.
	// - Clamp the speed change before putting the new speed in vel2
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	// Retrieve the current position as it relates to the grid, and the grid cell.
	glm::vec3 currentPosition = pos[index];
	glm::vec3 cellCenter = currentPosition + gridMin + (glm::vec3(0.5f, 0.5f, 0.5f) * cellWidth);
	glm::vec3 gridCoordinates = glm::floor((currentPosition - gridMin) * inverseCellWidth);

	// Figure out which octant of the cell the boid is in; two options along each of x, y, and z.
	// These options determine the neighboring 8 cells to check so that's encoded here.
	glm::ivec3 octant = glm::ivec3(gridCoordinates.x - 1, gridCoordinates.y - 1, gridCoordinates.z - 1);
	if (currentPosition.x > cellCenter.x) {
		++octant.x;
	}
	if (currentPosition.y > cellCenter.y) {
		++octant.y;
	}
	if (currentPosition.z > cellCenter.z) {
		++octant.z;
	}

	// Fields that we need to know about the selected boid.
	glm::vec3 currentVelocity = vel1[index];
	float perceivedMassNeighbors = 0.0f;
	glm::vec3 perceivedCenterTotal = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceivedCenterVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separationVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
	float perceivedVelocityNeighbors = 0.0f;
	glm::vec3 perceivedNeighborVelocityTotal = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceivedNeighborVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

	// Check the neighboring 8 cells
	for (int i = 0; i < 2; ++i) {
		for (int j = 0; j < 2; ++j) {
			for (int k = 0; k < 2; ++k) {

				// Restrict the set of neighbors to stay within the bounds of the grid.
				int neighborIndex = gridIndex3Dto1D(glm::clamp(octant.x + i, 0, gridResolution - 1), glm::clamp(octant.y + j, 0, gridResolution - 1),
					glm::clamp(octant.z + k, 0, gridResolution - 1), gridResolution);

				// Simulate these neighboring cell's boids.
				for (int boidIndex = gridCellStartIndices[neighborIndex]; boidIndex < gridCellEndIndices[neighborIndex] + 1; ++boidIndex) {
					if (boidIndex != -1 && boidIndex != index) {
						glm::vec3 neighborPosition = pos[index];
						glm::vec3 neighborVelocity = vel1[index];
						float distanceNeighbor = glm::distance(currentPosition, neighborPosition);

						// Find neighbors which factor into the perceived center of mass.
						if (distanceNeighbor < rule1Distance) {
							++perceivedMassNeighbors;
							perceivedCenterTotal += neighborPosition;
						}

						// Maintain separation.
						if (distanceNeighbor < rule2Distance) {
							separationVelocity -= (neighborPosition - currentPosition);
						}

						// Follow other boids.
						if (distanceNeighbor < rule3Distance) {
							++perceivedVelocityNeighbors;
							perceivedNeighborVelocityTotal += neighborVelocity;
						}
					}
				}
			}
		}
	}

	// Find the average of the perceived mass total for the center, then convert to velocity.
	if (perceivedMassNeighbors > 0) {
		perceivedCenterVelocity = ((perceivedCenterTotal / perceivedMassNeighbors) - currentPosition) * rule1Scale;
	}

	// Scale the separation velocity.
	separationVelocity *= rule2Scale;

	// Find the average of the perceived velocity total.
	if (perceivedVelocityNeighbors > 0) {
		perceivedNeighborVelocity = (perceivedNeighborVelocityTotal / perceivedVelocityNeighbors) * rule3Scale;
	}

	// The current velocity is added because this function doesn't actually compute the velocity change.
	// It's supposed to compute the new velocity instead.
	glm::vec3 newVelocity = (currentVelocity + perceivedCenterVelocity + separationVelocity + perceivedNeighborVelocity);
	if (glm::length(newVelocity) > maxSpeed) {
		newVelocity = (maxSpeed * glm::normalize(newVelocity));
	}
	vel2[index] = newVelocity;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	// TODO-1.2 ping-pong the velocity buffers
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
	kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
	dev_vel1 = dev_vel2;
}

void Boids::stepSimulationScatteredGrid(float dt) {
	// TODO-2.1
	// Uniform Grid Neighbor search using Thrust sort.
	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	// - Perform velocity updates using neighbor search
	// - Update positions
	// - Ping-pong buffers as needed
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// Sort based on the example given below as
	// LOOK-2.1 Example for using thrust::sort_by_key
	// Had this backwards the first time--swapping particleGrid and particleArray causes a crash.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// Initialize the start and end indices to invalid sentinel values.
	// This kernel expects to be passed the number of grid cells for the index buffers.
	dim3 gridCellBlocks((gridCellCount + blockSize - 1) / blockSize);
	kernResetIntBuffer<<<gridCellBlocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer<<<gridCellBlocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

	// Mark the divisions between grid cells and do the velocity update like the naive kernel.
	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	dev_vel1 = dev_vel2;
}

// Use a custom kernel to rearrange the array index buffer and make particle data coherent.
__global__ void kernMakeParticlesCoherentAgain(int numObjects, glm::vec3* dev_pos, 
	glm::vec3* dev_vel, glm::vec3* dev_coherentPos, glm::vec3* dev_coherentVel, 
	int* dev_particleArrayIndices) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= numObjects) {
		return;
	}

	// This is what we need to do to let kernUpdateVelNeighborSearchCoherent use its index 
	// directly, instead of needing to be passed dev_particleArrayIndices.
	int particleIndex = dev_particleArrayIndices[index];
	dev_coherentPos[index] = dev_pos[particleIndex];
	dev_coherentVel[index] = dev_vel[particleIndex];
}


void Boids::stepSimulationCoherentGrid(float dt) {
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	// In Parallel:
	// - Label each particle with its array index as well as its grid index.
	//   Use 2x width grids
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	// - Perform velocity updates using neighbor search
	// - Update positions
	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	
	// Sort based on the example given below as
	// LOOK-2.1 Example for using thrust::sort_by_key
	// Had this backwards the first time--swapping particleGrid and particleArray causes a crash.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// Initialize the start and end indices to invalid sentinel values.
	// This kernel expects to be passed the number of grid cells for the index buffers.
	dim3 gridCellBlocks((gridCellCount + blockSize - 1) / blockSize);
	kernResetIntBuffer<<<gridCellBlocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer<<<gridCellBlocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

	// Use a custom kernel to rearrange the array index buffer and make particle data coherent.
	kernMakeParticlesCoherentAgain << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, dev_vel1, dev_coherentPos, dev_coherentVel, dev_particleArrayIndices);

	// Mark the divisions between grid cells and do the velocity update like the naive kernel.
	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos, dev_vel1, dev_vel2);
	dev_vel1 = dev_vel2;
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_coherentPos);
  hipFree(dev_coherentVel);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
