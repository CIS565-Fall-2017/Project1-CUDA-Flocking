#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_vel1_shuffled;
glm::vec3 *dev_pos_shuffled;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount; //how many cells are there in total
int gridSideCount; //how many cells on one side of unit grid
float gridCellWidth; //side length of each grid cell
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;//Question: why halfSideCount?
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;
  std::cout << "gridMinimum is: " << gridMinimum.x << " " << gridMinimum.y << " " << gridMinimum.z << std::endl;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");
  hipDeviceSynchronize();

  hipMalloc((void**)&dev_vel1_shuffled, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1_shuffled failed!");

  hipMalloc((void**)&dev_pos_shuffled, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_shuffled failed!");
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	glm::vec3 velocityChange = glm::vec3(0);
	glm::vec3 perceived_center = glm::vec3(0);
	int count = 0;
	for (int i = 0; i < N; i++) {
		if (i != iSelf && glm::length(pos[i] - pos[iSelf]) < rule1Distance) {
			perceived_center += pos[i];
			count++;
		}
	}
	if (count > 0) {
		perceived_center /= count;
		velocityChange += (perceived_center - pos[iSelf]) * rule1Scale;
	}

  // Rule 2: boids try to stay a distance d away from each other
	glm::vec3 c(0,0,0);
	for (int i = 0; i < N; i++) {
		if (i != iSelf && glm::length(pos[i] - pos[iSelf]) < rule2Distance) {
			c -= (pos[i] - pos[iSelf]);
		}
	}
	velocityChange += c*rule2Scale;
  // Rule 3: boids try to match the speed of surrounding boids
	count = 0;
	glm::vec3 perceived_velocity = glm::vec3(0.f);
	for (int i = 0; i < N; i++) {
		if (pos[i] != pos[iSelf] && glm::length(pos[i] - pos[iSelf]) < rule3Distance){
			perceived_velocity += vel[i];
			count++;
		}
	}
	if (count > 0) {
		perceived_velocity /= count;
	}	
	velocityChange += perceived_velocity*rule3Scale;
	return vel[iSelf]+velocityChange;
  //return glm::vec3(0.0f, 0.0f, 0.0f);
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 newVel = computeVelocityChange(N, index, pos, vel1);
	glm::vec3 newVelFinal = newVel;
	
  // Clamp the speed
	if (glm::length(newVel) > maxSpeed) {
		newVelFinal = glm::normalize(newVel);
	}
  // Record the new velocity into vel2. Question: why NOT vel1?
	//Because we still need current state vel1 to determine other boid velocity change
	vel2[index] = newVelFinal;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
	if (x < 0 || x >= gridResolution) {
		return -1;
	}
	if (y < 0 || y >= gridResolution) {
		return -1;
	}
	if (z < 0 || z >= gridResolution) {
		return -1;
	}
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int numObjects, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
	int index = threadIdx.x + (blockIdx.x * blockDim.x); //get the parallel thread Id
	if (index >= numObjects) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	int xIndex = (thisPos.x-gridMin.x)*inverseCellWidth;
	int yIndex = (thisPos.y-gridMin.y)*inverseCellWidth;
	int zIndex = (thisPos.z-gridMin.z)*inverseCellWidth;
	gridIndices[index] = gridIndex3Dto1D(xIndex, yIndex, zIndex, gridResolution);

    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
	indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int cellNum, int objNum, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.	
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
	int cellIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (cellIndex >= cellNum) {
		return;
	}
	gridCellStartIndices[cellIndex] = -1;
	gridCellEndIndices[cellIndex] = -1;
	bool foundCellIndex = false;
	int i = 0;
	for (i = 0; i < objNum; i++) {		
		if (particleGridIndices[i] == cellIndex && !foundCellIndex) {
			gridCellStartIndices[cellIndex] = i;
			foundCellIndex = true;
		}
		if (particleGridIndices[i] != cellIndex && foundCellIndex) {
			gridCellEndIndices[cellIndex] = i-1;
			break;
		}
		if (particleGridIndices[i] > cellIndex) {
			//Sorted particleGridIndices, so if it is bigger, then it means we are not gonna find it
			break;
		}
	}
	if (i == objNum && foundCellIndex) {
		gridCellEndIndices[cellIndex] = i - 1;
	}
}

//Added by Yuxin, may not be correct due to the division number should be all neighboring particles instead of particles in one neighboring cell
__device__ glm::vec3 computeVelocityChangeNeighborSearchScattered(int iSelf, int neighborCellIndex, const glm::vec3 *pos, const glm::vec3 *vel,
	const int *gridCellStartIndices, const int *gridCellEndIndices, const int*particleArrayIndices) {
	// - For each cell, read the start/end indices in the boid pointer array.
	// - Access each boid in the cell and compute velocity change from
	//   the boids rules, if this boid is within the neighborhood distance.
	glm::vec3 velocityChange = glm::vec3(0);
	glm::vec3 perceived_center = glm::vec3(0);
	glm::vec3 perceived_velocity = glm::vec3(0.f);
	glm::vec3 c(0, 0, 0);
	int perceivedCenterCount = 0;
	int surroundSpeedCount = 0;
	int startIndex = gridCellStartIndices[neighborCellIndex];
	int endIndex = gridCellEndIndices[neighborCellIndex];
	if (startIndex != -1 && endIndex != -1) {
		for (int neighberIndex = startIndex; neighberIndex <= endIndex; neighberIndex++) {
			int particleIndex = particleArrayIndices[neighberIndex];
			// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
			if (particleIndex != iSelf && glm::length(pos[particleIndex] - pos[iSelf]) < rule1Distance) {
				perceived_center += pos[particleIndex];
				perceivedCenterCount++;
			}
			// Rule 2: boids try to stay a distance d away from each other			
			if (particleIndex != iSelf && glm::length(pos[particleIndex] - pos[iSelf]) < rule2Distance) {
				c -= (pos[particleIndex] - pos[iSelf]);
			}
			// Rule 3: boids try to match the speed of surrounding boids
			if (pos[particleIndex] != pos[iSelf] && glm::length(pos[particleIndex] - pos[iSelf]) < rule3Distance) {
				perceived_velocity += vel[particleIndex];
				surroundSpeedCount++;
			}
		}

		if (perceivedCenterCount > 0) {
			perceived_center /= perceivedCenterCount;
			velocityChange += (perceived_center - pos[iSelf]) * rule1Scale;
		}
		velocityChange += c*rule2Scale;
		if (surroundSpeedCount > 0) {
			perceived_velocity /= surroundSpeedCount;
		}
		velocityChange += perceived_velocity*rule3Scale;
	}
	return velocityChange;
}


__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
	int index = threadIdx.x + (blockIdx.x * blockDim.x); //get the parallel thread Id
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	int xIndex = (thisPos.x-gridMin.x)*inverseCellWidth;
	int yIndex = (thisPos.y-gridMin.y)*inverseCellWidth;
	int zIndex = (thisPos.z-gridMin.z)*inverseCellWidth;
  // - Identify which cells may contain neighbors. This isn't always 8. ??Questions which 8 cells??
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
 
	//Searching for 27 neighbors
	glm::vec3 velocityChange = glm::vec3(0);
	glm::vec3 perceived_center = glm::vec3(0);
	glm::vec3 perceived_velocity = glm::vec3(0.f);
	glm::vec3 c(0, 0, 0);
	int perceivedCenterCount = 0;
	int surroundSpeedCount = 0;
	int neighborCellIndex = -1;	
	for (int xNeighbor = xIndex - 1; xNeighbor <= xIndex + 1; xNeighbor++) {
		for (int yNeibor = yIndex - 1; yNeibor <= yIndex + 1; yNeibor++) {
			for (int zNeibor = zIndex - 1; zNeibor <= zIndex + 1; zNeibor++) {
				neighborCellIndex = gridIndex3Dto1D(xNeighbor, yNeibor, zNeibor, gridResolution);
				if (neighborCellIndex >= 0) {
					//velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, 
														//pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);										
					int startIndex = gridCellStartIndices[neighborCellIndex];
					int endIndex = gridCellEndIndices[neighborCellIndex];
					if (startIndex != -1 && endIndex != -1) {
						for (int neighberIndex = startIndex; neighberIndex <= endIndex; neighberIndex++) {
							int particleIndex = particleArrayIndices[neighberIndex];
							// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
							if (particleIndex != index && glm::length(pos[particleIndex] - thisPos) < rule1Distance) {
								perceived_center += pos[particleIndex];
								perceivedCenterCount++;
							}
							// Rule 2: boids try to stay a distance d away from each other			
							if (particleIndex != index && glm::length(pos[particleIndex] - thisPos) < rule2Distance) {
								c -= (pos[particleIndex] - thisPos);
							}
							// Rule 3: boids try to match the speed of surrounding boids
							if (particleIndex != index && glm::length(pos[particleIndex] - thisPos) < rule3Distance) {
								perceived_velocity += vel1[particleIndex];
								surroundSpeedCount++;
							}
						}
					}
				}				
			}
		}
	}
	if (perceivedCenterCount > 0) {
		perceived_center /= perceivedCenterCount;
		velocityChange += (perceived_center - thisPos) * rule1Scale;
	}
	velocityChange += c*rule2Scale;
	if (surroundSpeedCount > 0) {
		perceived_velocity /= surroundSpeedCount;
	}
	velocityChange += perceived_velocity*rule3Scale;

	/* searching for 8 neighbors, question: how to implement???
	float xPortion = (thisPos.x - cellWidth*xIndex) / cellWidth;
	float yPortion = (thisPos.y - cellWidth*yIndex) / cellWidth;
	float zPortion = (thisPos.z - cellWidth*zIndex) / cellWidth;
	bool leftX = xPortion < 0.5 ? true : false;
	bool lowerY = yPortion < 0.5 ? true : false;
	bool frontZ = zPortion < 0.5 ? true : false;
	if (leftX) {	
		//left X
		if (xIndex > 0) {
			neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex, zIndex, gridResolution);
			velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
			if (lowerY) {
				//left X + lower Y
				if (yIndex > 0) {
					neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex-1, zIndex, gridResolution);
					velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
					neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex - 1, zIndex, gridResolution);
					velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
					if (frontZ) {
						//left X + lower Y + front Z
						if (zIndex > 0) {
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex, zIndex-1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, 
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex-1, yIndex - 1, zIndex-1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, 
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex - 1, zIndex-1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex-1, yIndex, zIndex - 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
						}
					}
					else {
						//left X + lower Y + back Z
						if (zIndex < gridSideCount - 1) {
							neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex - 1, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex - 1, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
						}

					}
				}
			}
			else { 
				//left X + upper Y
				if (yIndex < gridSideCount - 1) {
					neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex + 1, zIndex, gridResolution);
					velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
					neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex + 1, zIndex, gridResolution);
					velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);

					if (frontZ) {
						//left X + upper Y + front Z
						if (zIndex > 0) {
							neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex + 1, zIndex - 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex + 1, zIndex - 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex-1, yIndex, zIndex - 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex, zIndex - 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
						}
					}
					else {
						//left X + upper Y + back Z
						if (zIndex < gridSideCount - 1) {
							neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex + 1, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex + 1, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex - 1, yIndex, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
							neighborCellIndex = gridIndex3Dto1D(xIndex, yIndex, zIndex + 1, gridResolution);
							velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, pos, vel1,
								gridCellStartIndices, gridCellEndIndices, particleArrayIndices);
						}

					}
				}
			}
		}
	}*/
  
	// - Clamp the speed change before putting the new speed in vel2
	//check which portion of the cell is the particle in, there are 8 portions each cell
	//on xdirection
	glm::vec3 newVel = vel1[index] + velocityChange;
	glm::vec3 finalNewVel = newVel;
	if (glm::length(newVel) > maxSpeed) {
		finalNewVel = glm::normalize(newVel);
	}
	vel2[index] = finalNewVel;
}

//Added by Yuxin, rearrange the position and velocity data to match sorted particle cell indice
__global__ void kernShufflePosVel(int N, int *particleArrayIndices, glm::vec3 *pos, glm::vec3 *pos_shuffled, 
	glm::vec3 *vel1, glm::vec3 *vel1_shuffled) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x); //get the parallel thread Id
	if (index >= N) {
		return;
	}
	int particleIndex_shuffled = particleArrayIndices[index];
	pos_shuffled[index] = pos[particleIndex_shuffled];
	vel1_shuffled[index] = vel1[particleIndex_shuffled];
}


__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos_shuffled, glm::vec3 *vel1_shuffled, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
	int index = threadIdx.x + (blockIdx.x * blockDim.x); //get the parallel thread Id
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos_shuffled[index];
	int xIndex = (thisPos.x - gridMin.x)*inverseCellWidth;
	int yIndex = (thisPos.y - gridMin.y)*inverseCellWidth;
	int zIndex = (thisPos.z - gridMin.z)*inverseCellWidth;
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  //Searching for 27 neighbors
	glm::vec3 velocityChange = glm::vec3(0);
	glm::vec3 perceived_center = glm::vec3(0);
	glm::vec3 perceived_velocity = glm::vec3(0.f);
	glm::vec3 c(0, 0, 0);
	int perceivedCenterCount = 0;
	int surroundSpeedCount = 0;
	int neighborCellIndex = -1;
	for (int xNeighbor = xIndex - 1; xNeighbor <= xIndex + 1; xNeighbor++) {
		for (int yNeibor = yIndex - 1; yNeibor <= yIndex + 1; yNeibor++) {
			for (int zNeibor = zIndex - 1; zNeibor <= zIndex + 1; zNeibor++) {
				neighborCellIndex = gridIndex3Dto1D(xNeighbor, yNeibor, zNeibor, gridResolution);
				if (neighborCellIndex >= 0) {
					//velocityChange += computeVelocityChangeNeighborSearchScattered(index, neighborCellIndex, 
					//pos, vel1, gridCellStartIndices, gridCellEndIndices, particleArrayIndices);										
					int startIndex = gridCellStartIndices[neighborCellIndex];
					int endIndex = gridCellEndIndices[neighborCellIndex];
					if (startIndex != -1 && endIndex != -1) {
						for (int neighberIndex = startIndex; neighberIndex <= endIndex; neighberIndex++) {
							// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
							if (neighberIndex != index && glm::length(pos_shuffled[neighberIndex] - thisPos) < rule1Distance) {
								perceived_center += pos_shuffled[neighberIndex];
								perceivedCenterCount++;
							}
							// Rule 2: boids try to stay a distance d away from each other			
							if (neighberIndex != index && glm::length(pos_shuffled[neighberIndex] - thisPos) < rule2Distance) {
								c -= (pos_shuffled[neighberIndex] - thisPos);
							}
							// Rule 3: boids try to match the speed of surrounding boids
							if (neighberIndex != index &&
								glm::length(pos_shuffled[neighberIndex] - thisPos) < rule3Distance) {
								perceived_velocity += vel1_shuffled[neighberIndex];
								surroundSpeedCount++;
							}
						}
					}
				}
			}
		}
	}
	if (perceivedCenterCount > 0) {
		perceived_center /= perceivedCenterCount;
		velocityChange += (perceived_center - pos_shuffled[index]) * rule1Scale;
	}
	velocityChange += c*rule2Scale;
	if (surroundSpeedCount > 0) {
		perceived_velocity /= surroundSpeedCount;
	}
	velocityChange += perceived_velocity*rule3Scale;

  // - Clamp the speed change before putting the new speed in vel2
	glm::vec3 newVel = vel1_shuffled[index] + velocityChange;
	glm::vec3 finalNewVel = newVel;
	if (glm::length(newVel) > maxSpeed) {
		finalNewVel = glm::normalize(newVel);
	}
	vel2[index] = finalNewVel;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdateVelocityBruteForce <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");
  // TODO-1.2 ping-pong the velocity buffers
	hipMemcpy(dev_vel1, dev_vel2, numObjects* sizeof(glm::vec3),hipMemcpyDeviceToDevice);

}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	//Debug
	/*int *particleGridIndex = new int[numObjects];
	int *particleIndex = new int[numObjects];
	// How to copy data back to the CPU side from the GPU
	hipMemcpy(particleGridIndex, dev_particleGridIndices, sizeof(int) * numObjects, hipMemcpyDeviceToHost);
	hipMemcpy(particleIndex, dev_particleArrayIndices, sizeof(int) * numObjects, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < 100; i++) {
		std::cout << "  key: " << particleGridIndex[i];
		std::cout << " value: " << particleIndex[i] << std::endl;
	}

	// cleanup
	delete[] particleGridIndex;
	delete[] particleIndex;*/
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
	dim3 fullCellsPerGrid((gridCellCount + blockSize - 1) / blockSize);
	kernIdentifyCellStartEnd << <fullCellsPerGrid, blockSize >> >(gridCellCount, numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  // - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> >
	(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
  // - Update positions
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  // - Ping-pong buffers as needed
	hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:  

  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	dim3 fullCellsPerGrid((gridCellCount + blockSize - 1) / blockSize);
	kernIdentifyCellStartEnd << <fullCellsPerGrid, blockSize >> >(gridCellCount, numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	kernShufflePosVel << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleArrayIndices, dev_pos, dev_pos_shuffled,
		dev_vel1, dev_vel1_shuffled);
	
		// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> >
		(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
			dev_pos_shuffled, dev_vel1_shuffled, dev_vel2);
	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos_shuffled, dev_vel2);
	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

	/*glm::vec3* temp;
	temp = dev_pos;
	dev_pos = dev_pos_shuffled;
	dev_pos_shuffled = temp;

	temp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = dev_vel1;*/

	hipMemcpy(dev_pos, dev_pos_shuffled, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
	hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	//Debug
	/*glm::vec3 *particlePos = new glm::vec3[numObjects];
	glm::vec3 *particleVel = new glm::vec3[numObjects];
	// How to copy data back to the CPU side from the GPU
	hipMemcpy(particlePos, dev_pos, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToHost);
	hipMemcpy(particleVel, dev_vel1, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after update vel and pos in coherent grids: " << std::endl;
	for (int i = 0; i < 100; i++) {
		std::cout << " position: " << "["<< particlePos[i].x<<" "<< particlePos[i].y<<" "<< particlePos[i].z<<std::endl;
		std::cout << " velocity: " << "[" << particleVel[i].x << " " << particleVel[i].y << " " << particleVel[i].z << std::endl;
	}

	// cleanup
	delete[] particlePos;
	delete[] particleVel;*/
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_vel1_shuffled);
  hipFree(dev_pos_shuffled);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
