#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.

  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");


  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  hipDeviceSynchronize();

}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {

	glm::vec3 boidPos = pos[iSelf];

	glm::vec3 perceived_center = glm::vec3(0.0f,0.0f,0.0f);

	glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);

	glm::vec3 perceived_vel = glm::vec3(0.0f, 0.0f, 0.0f);


	int neighbors1 = 0;

	int neighbors3 = 0;

	for (int i = 0; i < N; i++) {

		if (i != iSelf) {

		glm::vec3 bPos = pos[i];

		float dist = glm::distance(bPos, boidPos);

		// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves

			if (dist < rule1Distance) {
				//if (iSelf == 64) {
				//	printf("neighbor1 %f %f %f \n", bPos.x, bPos.y, bPos.z);
				//}
				neighbors1++;
				perceived_center = perceived_center + bPos;
			}

		// Rule 2: boids try to stay a distance d away from each other

			if (dist < rule2Distance) {
				c = c - (bPos - boidPos);
			}

		// Rule 3: boids try to match the speed of surrounding boid

			if (dist < rule3Distance) {
				perceived_vel = perceived_vel + vel[i];
				neighbors3++;
			}
		}
	}

	glm::vec3 r1dv = glm::vec3(0.0f,0.0f,0.0f);
	if (neighbors1 > 0) {
		perceived_center /= (float)neighbors1;
		r1dv = (perceived_center - boidPos) * rule1Scale;
	}


	glm::vec3 r2dv = c * rule2Scale;


	glm::vec3 r3dv = glm::vec3(0.0f, 0.0f, 0.0f);
	if (neighbors3 > 0) {
		perceived_vel /= (float)neighbors3; 
		r3dv = perceived_vel * rule3Scale;
	}
	//if (iSelf == 64) {
	//	printf("%f %f %f \n", boidPos.x, boidPos.y, boidPos.z);
	//	printf("%f %f %f \n", r1dv.x, r1dv.y, r1dv.z);
	//	printf("%f %f %f \n", r2dv.x, r2dv.y, r2dv.z);
	//	printf("%f %f %f \n", r3dv.x, r3dv.y, r3dv.z);
	//}
	return r1dv + r2dv + r3dv + vel[iSelf];
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	glm::vec3 new_vel = computeVelocityChange(N, index, pos, vel1);
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
	if (glm::length(new_vel) > maxSpeed) {
		new_vel = glm::normalize(new_vel) * maxSpeed;
	}

	vel2[index] = new_vel;
  // Record the new velocity into vel2. Question: why NOT vel1?
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

	

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 gs_pos = glm::floor((pos[index] - gridMin) * inverseCellWidth);
	/*printf("%f", inverseCellWidth);
	printf("%i %f %f %f\n", index, pos[index].x, pos[index].y, pos[index].z);

	printf("%i %f %f %f\n", index, gs_pos.x, gs_pos.y, gs_pos.z);*/

	int gridIndex = gridIndex3Dto1D(gs_pos.x, gs_pos.y, gs_pos.z, gridResolution);

	indices[index] = index;
	gridIndices[index] = gridIndex;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	int gridIdx = particleGridIndices[index];
	if (index == 0) {
		gridCellStartIndices[gridIdx] = 0;
	}
	if (index == N - 1) {
		gridCellEndIndices[gridIdx] = N - 1;
	}
	if (index != 0 && gridIdx != particleGridIndices[index - 1]) {
		gridCellStartIndices[gridIdx] = index;
		//if (gridIdx == 8341) {
		//	printf("%i start %i\n", gridIdx, index);
		//}
	}
	if (index != N-1 && gridIdx != particleGridIndices[index + 1]) {
		gridCellEndIndices[gridIdx] = index;
		//if (gridIdx == 8341) {
		//	printf("%i end %i\n", gridIdx, index);
		//}
		//printf("%i end %i\n", gridIdx, index);
	}
	/*printf("%i %i\n", gridIdx, index);
	printf("%i start %i\n", gridIdx, gridCellStartIndices[gridIdx]);
	printf("%i end %i\n", gridIdx, gridCellEndIndices[gridIdx]);*/

  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	//if (index == 0) {
	//	int s = gridCellStartIndices[8341];
	//	int e = gridCellEndIndices[8341];
	//	printf("start %i end %i \n", s, e);
	//}
	//printf("ws_pos %i %f %f\n", gridResolution, inverseCellWidth, cellWidth);
	glm::vec3 ws_pos = pos[index];
	//printf("ws_pos %i %f %f %f\n", index, ws_pos.x, ws_pos.y, ws_pos.z);
	glm::vec3 ws_shift_pos = ws_pos - gridMin;
	//printf("ws_s_pos %i %f %f %f\n", index, ws_shift_pos.x, ws_shift_pos.y, ws_shift_pos.z);
	glm::vec3 gs_posf =(ws_shift_pos * inverseCellWidth);
	glm::vec3 gs_pos;
	gs_pos.x = (int)gs_posf.x;
	gs_pos.y = (int)gs_posf.y;
	gs_pos.z = (int)gs_posf.z;
	

	glm::vec3 quadf = (ws_shift_pos - (gs_pos * cellWidth)) - (cellWidth/2.0f);
	glm::vec3 quad(0, 0, 0);
	quad.x = (int)(quadf.x / fabsf(quadf.x));
	quad.y = (int)(quadf.y / fabsf(quadf.y));
	quad.z = (int)(quadf.z / fabsf(quadf.z));
	/*if (index == 64) {
		printf("gs_pos %i %f %f %f\n", index, gs_pos.x, gs_pos.y, gs_pos.z);
		printf("quad %i %f %f %f\n", index, quad.x, quad.y, quad.z);
	}*/

	//printf("quad_norm %i %i %i %i\n", index, quad.x, quad.y, quad.z);
	glm::vec3 boidPos = ws_pos;
	int neighbors1 = 0;
	int neighbors3 = 0;
	glm::vec3 c(0.0f, 0.0f, 0.0f);
	glm::vec3 perceived_center(0.0f, 0.0f, 0.0f);
	glm::vec3 perceived_vel(0.0f, 0.0f, 0.0f);
	int gridRes = gridResolution;
	for (int i = 0; i < 2; i++) {
		int xn = gs_pos.x + (i*quad.x);
		if (xn >= gridRes || xn < 0) { continue; }
		for (int j = 0; j < 2; j++) {
			int yn = gs_pos.y + (j*quad.y);
			if (yn >= gridRes || yn < 0) { continue; }
			for (int k = 0; k < 2; k++) {
				int zn = gs_pos.z + (k*quad.z);
				if (zn >= gridRes || zn < 0) { continue; }
				glm::vec3 check_grid_cell(xn, yn, zn);
				//printf("quad_norm %i %f %f %f\n", index, check_grid_cell.x, check_grid_cell.y, check_grid_cell.z);
				int grid_idx = gridIndex3Dto1D(check_grid_cell.x, check_grid_cell.y, check_grid_cell.z, gridRes);
				int start_idx = gridCellStartIndices[grid_idx];
				int end_idx = gridCellEndIndices[grid_idx];
				//printf("start %i end %i \n", start_idx, end_idx);
				//if (grid_idx == 8341) {
				//	printf("start\n");
				//}
				if (start_idx == -1) {
					continue;
				}
				for (int idx = start_idx; idx < end_idx + 1; idx++) {
					

						//if (grid_idx == 8341) {
						//	printf("%i %i %i \n", grid_idx, index, idx);
						//}
						int boid_id = particleArrayIndices[idx];
						glm::vec3 bPos = pos[boid_id];

						float dist = glm::distance(bPos, boidPos);

						// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
					if (dist > 0) {
						if (dist < rule1Distance) {
							if (index == 64) {
								//printf("neighbor1 %f %f %f \n", bPos.x, bPos.y, bPos.z);
							}
							neighbors1++;
							perceived_center = perceived_center + bPos;
						}

						// Rule 2: boids try to stay a distance d away from each other

						if (dist < rule2Distance) {
							c = c - (bPos - boidPos);
						}

						// Rule 3: boids try to match the speed of surrounding boid

						if (dist < rule3Distance) {
							perceived_vel = perceived_vel + vel1[boid_id];
							neighbors3++;
						}
					}
				}
			}
		}
	}
	glm::vec3 r1dv = glm::vec3(0.0f, 0.0f, 0.0f);
	if (neighbors1 > 0) {
		perceived_center /= (float)neighbors1;
		r1dv = (perceived_center - boidPos) * rule1Scale;
	}

	glm::vec3 r2dv = c * rule2Scale;

	glm::vec3 r3dv = glm::vec3(0.0f, 0.0f, 0.0f);
	if (neighbors3 > 0) {
		perceived_vel /= (float)neighbors3;
		r3dv = perceived_vel * rule3Scale;
	}

	glm::vec3 new_vel = r1dv + r2dv + r3dv + vel1[index];
	if (glm::length(new_vel) > maxSpeed) {
		new_vel = glm::normalize(new_vel) * maxSpeed;
	}
	//if (index == 64) {
	//	printf("%f %f %f \n", ws_pos.x, ws_pos.y, ws_pos.z);
	//	printf("%f %f %f \n", r1dv.x, r1dv.y, r1dv.z);
	//	printf("%f %f %f \n", r2dv.x, r2dv.y, r2dv.z);
	//	printf("%f %f %f \n", r3dv.x, r3dv.y, r3dv.z);
	//}
	vel2[index] = new_vel;

  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	int N = numObjects;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
	kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> >(N, dev_pos, dev_vel1, dev_vel2);
	kernUpdatePos <<<fullBlocksPerGrid, blockSize >>>(N, dt, dev_pos, dev_vel2);


	checkCUDAErrorWithLine("stepSimulationNaive failed!");

	hipDeviceSynchronize();

	glm::vec3* temp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp;

  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.

	int N = numObjects;

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (N, dev_particleArrayIndices, 0);
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (N, dev_particleGridIndices, 0);

	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);


	dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
	dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + N, dev_thrust_particleArrayIndices);


	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	int grid_size = gridSideCount * gridSideCount  *gridSideCount;

	dim3 fullBlocksPerGrid_GRID((grid_size + blockSize - 1) / blockSize);
	kernResetIntBuffer << <fullBlocksPerGrid_GRID, blockSize >> > (grid_size, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGrid_GRID, blockSize >> > (grid_size, dev_gridCellEndIndices, -1);

	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(N,dev_particleGridIndices,dev_gridCellStartIndices,dev_gridCellEndIndices);
	

	// - Perform velocity updates using neighbor search
	
	kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (N, gridSideCount, gridMinimum, gridInverseCellWidth,
		gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	
	//kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> >(N, dev_pos, dev_vel1, dev_vel2);

	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(N, dt, dev_pos, dev_vel2);

	checkCUDAErrorWithLine("stepSimulationNaive failed!");

	hipDeviceSynchronize();

	glm::vec3* temp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp;

  // - Ping-pong buffers as needed
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);


  hipFree(dev_particleGridIndices);
  hipFree(dev_particleArrayIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
