#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_new_pos;
glm::vec3 *dev_new_vel1;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  // Allocate dev_particleArrayIndices in device memory
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  // Allocate dev_particleGridIndices in device memory
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  // Allocate dev_gridCellStartIndices in device memory
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  // Allocate dev_gridCellEndIndices in device memory
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  //Allocate new position and velocity buffers in device memory for coherent implementation
  hipMalloc((void**)&dev_new_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_new_pos failed!");
  hipMalloc((void**)&dev_new_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_new_pos failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
	// Final velocity of the boid
	glm::vec3 finalVelocity(0.0f, 0.0f, 0.0f);

	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	glm::vec3 perceivedCenter(0.0f, 0.0f, 0.0f);
	int centerInfluencerCount = 0;
	// Rule 2: boids try to stay a distance d away from each other
	glm::vec3 seperation(0.0f, 0.0f, 0.0f);
	// Rule 3: boids try to match the speed of surrounding boids
	glm::vec3 perceivedVelocity(0.0f, 0.0f, 0.0f);
	int velocityInfluencerCount = 0;

	for (int i = 0; i < N; ++i) {
		if (i == iSelf) {
			continue;
		}
		float distance = glm::distance(pos[i], pos[iSelf]);

		// Rule 1 cohesion
		if (distance < rule1Distance) {
			perceivedCenter += pos[i];
			centerInfluencerCount++;
		}

		// Rule 2 separation
		if (distance < rule2Distance) {
			seperation -= (pos[i] - pos[iSelf]);
		}

		// Rule 3 alignment
		if (distance < rule3Distance) {
			perceivedVelocity += vel[i];
			velocityInfluencerCount++;
		}
	}

	if (centerInfluencerCount > 0) {
		perceivedCenter /= centerInfluencerCount;
		finalVelocity += (perceivedCenter - pos[iSelf]) * rule1Scale;
	}
	
	if (velocityInfluencerCount > 0) {
		perceivedVelocity /= velocityInfluencerCount;
		finalVelocity += perceivedVelocity * rule3Scale;
	}

	finalVelocity += seperation * rule2Scale;

	return finalVelocity;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 tempVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);
	float boidSpeed = glm::length(tempVel);
	if (boidSpeed > maxSpeed) {
		tempVel = (tempVel / boidSpeed) * maxSpeed;
	}
	vel2[index] = tempVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < N) {
		// Filling the buffer dev_particleArrayIndices with index position of the boid
		indices[index] = index;

		// Find out the cell in which the boid exists
		// Fill the dev_particleGridIndices buffer with the value of the cell index
		glm::ivec3 gridIdx = (pos[index] - gridMin) * inverseCellWidth;
		gridIndices[index] = gridIndex3Dto1D(gridIdx.x, gridIdx.y, gridIdx.z, gridResolution);
	}
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	int nextIndex = index + 1;

	if (index < N) {
		int valAtIndex = particleGridIndices[index];
		int valAtNextIndex = particleGridIndices[nextIndex];
		if (index == 0) {
			if (valAtIndex == valAtNextIndex) {
				gridCellStartIndices[valAtIndex] = index;
			}
			else {
				gridCellStartIndices[valAtIndex] = index;
				gridCellEndIndices[valAtIndex] = index;

				gridCellStartIndices[valAtNextIndex] = nextIndex;
			}
		} 
		else if (nextIndex == N) {
			gridCellEndIndices[valAtIndex] = index;
		}
		else {
			if (valAtIndex != valAtNextIndex) {
				gridCellEndIndices[valAtIndex] = index;
				gridCellStartIndices[valAtNextIndex] = nextIndex;
			}
		}
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) {
		return;
	}

	// Final velocity of the boid
	glm::vec3 finalVelocity(0.0f, 0.0f, 0.0f);

	// Rule 1: Cohesion
	glm::vec3 perceivedCenter(0.0f, 0.0f, 0.0f);
	int centerInfluencerCount = 0;
	// Rule 2: Seperation
	glm::vec3 seperation(0.0f, 0.0f, 0.0f);
	// Rule 3: Alignment
	glm::vec3 perceivedVelocity(0.0f, 0.0f, 0.0f);
	int velocityInfluencerCount = 0;

	// - Identify the grid cell that this particle/boid is in
	glm::vec3 boidPos = pos[index];
	glm::ivec3 boidGridIndex = (boidPos - gridMin) * inverseCellWidth;
	int boidCellIndex = gridIndex3Dto1D(boidGridIndex.x, boidGridIndex.y, boidGridIndex.z, gridResolution);

	// - Identify which cells may contain neighbors. This isn't always 8.
	float neighbourhoodDistance = imax(imax(rule1Distance, rule2Distance), rule3Distance);

	// - For each cell, read the start/end indices in the boid pointer array.
	// - Access each boid in the cell and compute velocity change from	
	//   the boids rules, if this boid is within the neighborhood distance.
	for (int i = -1; i < 2; ++i) {
		for (int j = -1; j < 2; ++j) {
			for (int k = -1; k < 2; ++k) {
				int x = boidGridIndex.x + i;
				int y = boidGridIndex.y + j;
				int z = boidGridIndex.z + k;

				// - Handeling Edge Case
				x = imax(x, 0);
				y = imax(y, 0);
				z = imax(z, 0);

				x = imin(x, gridResolution - 1);
				y = imin(y, gridResolution - 1);
				z = imin(z, gridResolution - 1);
				
				// - Index of the neighbouring grid cell
				int neighbourGridCellIndex = gridIndex3Dto1D(x, y, z, gridResolution);

				if (gridCellStartIndices[neighbourGridCellIndex] != -1) {
					// - Loop through the boids in this grid cell
					// - Update the velocity of the current boid based on the neighbourhood distance
					for (int l = gridCellStartIndices[neighbourGridCellIndex]; l <= gridCellEndIndices[neighbourGridCellIndex]; ++l) {
						int boidIndex = particleArrayIndices[l];
						if (boidIndex != index) {
							float distance = glm::distance(boidPos, pos[boidIndex]);

							// - Rule 1 cohesion
							if (distance < rule1Distance) {
								perceivedCenter += pos[boidIndex];
								centerInfluencerCount++;
							}

							// - Rule 2 separation
							if (distance < rule2Distance) {
								seperation -= (pos[boidIndex] - boidPos);
							}

							// - Rule 3 alignment
							if (distance < rule3Distance) {
								perceivedVelocity += vel1[boidIndex];
								velocityInfluencerCount++;
							}
						}
					}
				}
			}
		}
	}

	if (centerInfluencerCount > 0) {
		perceivedCenter /= centerInfluencerCount;
		finalVelocity += (perceivedCenter - pos[index]) * rule1Scale;
	}

	if (velocityInfluencerCount > 0) {
		perceivedVelocity /= velocityInfluencerCount;
		finalVelocity += perceivedVelocity * rule3Scale;
	}

	finalVelocity += seperation * rule2Scale;
	finalVelocity += vel1[index];

	// - Clamp the speed change before putting the new speed in vel2
	float boidSpeed = glm::length(finalVelocity);
	if (boidSpeed > maxSpeed) {
		finalVelocity = (finalVelocity / boidSpeed) * maxSpeed;
	}
	vel2[index] = finalVelocity;
}

__global__ void kernSwapPosVel(int N, int *particleArrayIndices, glm::vec3 *new_pos, glm::vec3 *new_vel, glm::vec3 *pos, glm::vec3 *vel) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) {
		return;
	}

	int boidIndex = particleArrayIndices[index];
	new_pos[index] = pos[boidIndex];
	new_vel[index] = vel[boidIndex];
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2


	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) {
		return;
	}

	// Final velocity of the boid
	glm::vec3 finalVelocity(0.0f, 0.0f, 0.0f);

	// Rule 1: Cohesion
	glm::vec3 perceivedCenter(0.0f, 0.0f, 0.0f);
	int centerInfluencerCount = 0;
	// Rule 2: Seperation
	glm::vec3 seperation(0.0f, 0.0f, 0.0f);
	// Rule 3: Alignment
	glm::vec3 perceivedVelocity(0.0f, 0.0f, 0.0f);
	int velocityInfluencerCount = 0;

	// - Identify the grid cell that this particle/boid is in
	glm::vec3 boidPos = pos[index];
	glm::ivec3 boidGridIndex = (boidPos - gridMin) * inverseCellWidth;
	int boidCellIndex = gridIndex3Dto1D(boidGridIndex.x, boidGridIndex.y, boidGridIndex.z, gridResolution);

	// - Identify which cells may contain neighbors. This isn't always 8.
	float neighbourhoodDistance = imax(imax(rule1Distance, rule2Distance), rule3Distance);

	// - For each cell, read the start/end indices in the boid pointer array.
	// - Access each boid in the cell and compute velocity change from	
	//   the boids rules, if this boid is within the neighborhood distance.
	for (int i = -1; i < 2; ++i) {
		for (int j = -1; j < 2; ++j) {
			for (int k = -1; k < 2; ++k) {
				int x = boidGridIndex.x + i;
				int y = boidGridIndex.y + j;
				int z = boidGridIndex.z + k;

				// - Handeling Edge Case
				x = imax(x, 0);
				y = imax(y, 0);
				z = imax(z, 0);

				x = imin(x, gridResolution - 1);
				y = imin(y, gridResolution - 1);
				z = imin(z, gridResolution - 1);

				// - Index of the neighbouring grid cell
				int neighbourGridCellIndex = gridIndex3Dto1D(x, y, z, gridResolution);

				if (gridCellStartIndices[neighbourGridCellIndex] != -1) {
					// - Loop through the boids in this grid cell
					// - Update the velocity of the current boid based on the neighbourhood distance
					for (int boidIndex = gridCellStartIndices[neighbourGridCellIndex]; boidIndex <= gridCellEndIndices[neighbourGridCellIndex]; ++boidIndex) {
						if (boidIndex != index) {
							float distance = glm::distance(boidPos, pos[boidIndex]);

							// - Rule 1 cohesion
							if (distance < rule1Distance) {
								perceivedCenter += pos[boidIndex];
								centerInfluencerCount++;
							}

							// - Rule 2 separation
							if (distance < rule2Distance) {
								seperation -= (pos[boidIndex] - boidPos);
							}

							// - Rule 3 alignment
							if (distance < rule3Distance) {
								perceivedVelocity += vel1[boidIndex];
								velocityInfluencerCount++;
							}
						}
					}
				}
			}
		}
	}

	if (centerInfluencerCount > 0) {
		perceivedCenter /= centerInfluencerCount;
		finalVelocity += (perceivedCenter - pos[index]) * rule1Scale;
	}

	if (velocityInfluencerCount > 0) {
		perceivedVelocity /= velocityInfluencerCount;
		finalVelocity += perceivedVelocity * rule3Scale;
	}

	finalVelocity += seperation * rule2Scale;
	finalVelocity += vel1[index];

	// - Clamp the speed change before putting the new speed in vel2
	float boidSpeed = glm::length(finalVelocity);
	if (boidSpeed > maxSpeed) {
		finalVelocity = (finalVelocity / boidSpeed) * maxSpeed;
	}
	vel2[index] = finalVelocity;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	dim3 blockspergrid((numObjects + blockSize - 1) / blockSize);

  // todo-1.2 - use the kernels you wrote to step the simulation forward in time.
	kernUpdateVelocityBruteForce<<<blockspergrid , blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

	kernUpdatePos<<<blockspergrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

  // todo-1.2 ping-pong the velocity bufferss
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
	// TODO-2.1
	// Uniform Grid Neighbor search using Thrust sort.

	// - Setup the dimensions for the kernels
	dim3 fullBlocksPerGrid_GridCellDimension((gridCellCount + blockSize - 1) / blockSize);
	dim3 fullBlocksPerGrid_BoidDimension((numObjects + blockSize - 1) / blockSize);

	// - PreFill the start and end index arrays with value -1
	//   This value will be used to determine if there are boids in the cells
	kernResetIntBuffer <<<fullBlocksPerGrid_GridCellDimension, blockSize>>> (gridCellCount, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for start indices failed!");
	kernResetIntBuffer <<<fullBlocksPerGrid_GridCellDimension, blockSize>>> (gridCellCount, dev_gridCellEndIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for end indicesfailed!");

	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	kernComputeIndices <<<fullBlocksPerGrid_BoidDimension, blockSize>>> (numObjects,  gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	// - LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	checkCUDAErrorWithLine("thrust::sort_by_key failed!");

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd <<<fullBlocksPerGrid_BoidDimension, blockSize>>> (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered <<<fullBlocksPerGrid_BoidDimension, blockSize>>> (numObjects, gridSideCount, gridMinimum,
																						   gridInverseCellWidth, gridCellWidth,
																						   dev_gridCellStartIndices, dev_gridCellEndIndices,
																						   dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

	// - Update positions
	kernUpdatePos <<<fullBlocksPerGrid_BoidDimension, blockSize>>> (numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// - Ping-pong buffers as needed
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

  // - Setup the dimensions for the kernels
	dim3 fullBlocksPerGrid_GridCellDimension((gridCellCount + blockSize - 1) / blockSize);
	dim3 fullBlocksPerGrid_BoidDimension((numObjects + blockSize - 1) / blockSize);

	// - PreFill the start and end index arrays with value -1
	//   This value will be used to determine if there are boids in the cells
	kernResetIntBuffer <<<fullBlocksPerGrid_GridCellDimension, blockSize >>> (gridCellCount, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for start indices failed!");
	kernResetIntBuffer <<<fullBlocksPerGrid_GridCellDimension, blockSize >>> (gridCellCount, dev_gridCellEndIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for end indicesfailed!");

	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	kernComputeIndices <<<fullBlocksPerGrid_BoidDimension, blockSize >>> (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	// - LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	checkCUDAErrorWithLine("thrust::sort_by_key failed!");

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd <<<fullBlocksPerGrid_BoidDimension, blockSize >>> (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	kernSwapPosVel <<<fullBlocksPerGrid_BoidDimension, blockSize >>> (numObjects, dev_particleArrayIndices, dev_new_pos, dev_new_vel1, dev_pos, dev_vel1);

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent <<<fullBlocksPerGrid_BoidDimension, blockSize >>> (numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_new_pos, dev_new_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

	// - Update positions
	kernUpdatePos <<<fullBlocksPerGrid_BoidDimension, blockSize >>> (numObjects, dt, dev_new_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// - Ping-pong buffers as needed
	std::swap(dev_vel1, dev_vel2);
	std::swap(dev_new_pos, dev_pos);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_new_pos);
  hipFree(dev_new_vel1);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
