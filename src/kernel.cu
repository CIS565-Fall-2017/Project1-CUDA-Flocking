#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *coherent_pos;
glm::vec3 *coherent_vel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;	//num of cells in entire grid 
int gridSideCount;	//num of cells along one side in the grid
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&coherent_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc coherent_pos failed!");

  hipMalloc((void**)&coherent_vel, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc coherent_vel failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids

	glm::vec3 v1;
	glm::vec3 v2;
	glm::vec3 v3;

	int neighborCount1 = 0;
	int neighborCount2 = 0;

	for (int j = 0; j < N; j++)
	{
		//if (j == iSelf)	continue;

		float distance = glm::distance(pos[j], pos[iSelf]);

		if (j != iSelf)
		{
			if (distance < rule1Distance)
			{
				v1 += pos[j];
				neighborCount1++;
			}

			if (distance < rule2Distance)
			{
				v2 -= (pos[j] - pos[iSelf]);
			}

			if (distance < rule3Distance)
			{
				v3 += vel[j];
				neighborCount2++;
			}
		}//end if j!= iself
	}//end for loop

	if (neighborCount1 > 0)
	{
		v1 /= neighborCount1;
		v1 = (v1 - pos[iSelf]) * rule1Scale;
	}

	if (neighborCount2 > 0)
	{
		v3 /= neighborCount2;
		v3 *= rule3Scale;
	}

	v2 *= rule2Scale;

	return v1 + v2 + v3;
}

// --------------------------------------------------------------------------------------------

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	glm::vec3 totalVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);

	if (glm::length(totalVel) > maxSpeed)
	{
		totalVel = glm::normalize(totalVel) * maxSpeed;
	}

	vel2[index] = totalVel;
}

// --------------------------------------------------------------------------------------------

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// ===========================================================================================
/*
								COMPUTE INDICES FUNCTIONS
*/
// ===========================================================================================

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

// --------------------------------------------------------------------------------------------

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	//Need to do this b/c the simulation cube origin is (0,0,0), but bottom left corner of grid is (0,0,0). 
	glm::ivec3 particleGridPos = glm::round((pos[index] - gridMin) * inverseCellWidth);
	
	//[QUESTION] Do I need to clamp so that position doesn't produce an index out of range?
	//glm::clamp(particleGridPos, 0.0f, (float)(gridResolution - 1));

	int particleGridIdx = gridIndex3Dto1D(particleGridPos.x, particleGridPos.y, particleGridPos.z, gridResolution);

	indices[index] = index;
	gridIndices[index] = particleGridIdx;
}

// --------------------------------------------------------------------------------------------

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

// --------------------------------------------------------------------------------------------

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	//Edge cases:
	//Since you're always filling in the start for index + 1, you need to separately figure out start for first index
	//Since you're always filling the end for index, you need to separately figure out end for last index
	//Note: this is NOT ideal since this assumes there are always particles in the first and last grid cells
	if (index == 0)
	{
		gridCellStartIndices[particleGridIndices[index]] = 0;
	}

	else if (index == N - 1)
	{
		gridCellEndIndices[particleGridIndices[index]] = N - 1;
	}

	//Fill in the start of index + 1 and the end of index 
	//only checking in particleGridIndices where the values differ
	if (particleGridIndices[index] != particleGridIndices[index + 1])
	{
		gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;
		gridCellEndIndices[particleGridIndices[index]] = index;
	}

}

// --------------------------------------------------------------------------------------------

__global__ void kernComputeCoherentBuffers(int N, int *particleArrayIndices,
	const glm::vec3 *pos, const glm::vec3 *vel, 
	glm::vec3 *coher_pos, glm::vec3 *coher_vel)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	int particleIdx = particleArrayIndices[index];
	coher_pos[index] = pos[particleIdx];
	coher_vel[index] = vel[particleIdx];
}

// ===========================================================================================
/*
									UPDATE VELOCITY FUNCTIONS
*/
// ===========================================================================================

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	glm::ivec3 particleGridPos = glm::round((pos[index] - gridMin) * inverseCellWidth);

	//[QUESTION] Do I need to clamp? Here?
	//glm::clamp(particleGridPos, 0.0f, (float)(gridResolution - 1));

	//iterate through neighbors of current grid cell
	glm::vec3 v1 = glm::vec3(0.0f);
	glm::vec3 v2 = glm::vec3(0.0f);
	glm::vec3 v3 = glm::vec3(0.0f);
	int neighborCount1 = 0;
	int neighborCount2 = 0;

	glm::vec3 currParticlePos = pos[index];
	glm::vec3 currParticleVel = vel1[index];

	for (int i = -1; i < 1; i++) {
		for (int j = -1; j < 1; j++) {
			for (int k = -1; k < 1; k++) {

				//[QUESTION] Do I need to clamp?
				//separate these out and imax(x + i, 0) and imin(_x, gridResolution - 1)
				//int _x = imin(imax(particleGridPos.x + i, 0), gridResolution - 1);
				//int _y = imin(imax(particleGridPos.y + j, 0), gridResolution - 1);
				//int _z = imin(imax(particleGridPos.z + k, 0), gridResolution - 1);
				//int particleGridIdx = gridIndex3Dto1D(_x, _y, _z, gridResolution);
				int particleGridIdx = gridIndex3Dto1D(particleGridPos.x + i, particleGridPos.y + j, particleGridPos.z + k, gridResolution);

				int startIdx = gridCellStartIndices[particleGridIdx];
				int endIdx = gridCellEndIndices[particleGridIdx];

				if (startIdx != -1)	//shouldn't need to check endIdx b/c startIdx and endIdx should be -1 in the same places
				{
					for (int n = startIdx; n <= endIdx; n++)
					{
						//distance between current boid in for loop and this boid that's at index
						glm::vec3 currNeighboringParticlePos = pos[particleArrayIndices[n]];
						glm::vec3 currNeighboringParticleVel = vel1[particleArrayIndices[n]];

						float distance = glm::distance(currNeighboringParticlePos, currParticlePos);

						if (n != index)
						{
							if (distance < rule1Distance)
							{
								v1 += currNeighboringParticlePos;
								neighborCount1++;
							}

							if (distance < rule2Distance)
							{
								v2 -= (currNeighboringParticlePos - currParticlePos);
							}

							if (distance < rule3Distance)
							{
								v3 += currNeighboringParticleVel;
								neighborCount2++;
							}
						}//end if n!= index
					}//end for n
				}//end if startIdx != -1
			}//end for k
		}//end for j
	}//end for i


	if (neighborCount1 > 0)
	{
		v1 /= neighborCount1;
		v1 = (v1 - currParticlePos) * rule1Scale;
	}

	if (neighborCount2 > 0)
	{
		v3 /= neighborCount2;
		v3 *= rule3Scale;
	}

	v2 *= rule2Scale;

	glm::vec3 totalVel = currParticleVel + v1 + v2 + v3;

	//Clamp the speed change
	if (glm::length(totalVel) > maxSpeed)
	{
		totalVel = glm::normalize(totalVel) * maxSpeed;
	}

	vel2[index] = totalVel;
}

// --------------------------------------------------------------------------------------------

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

	//REWRITE THE FOLLOWING

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	} 

	glm::ivec3 particleGridPos = glm::round((pos[index] - gridMin) * inverseCellWidth);

	//[QUESTION] Do I need to clamp? Here?
	//glm::clamp(particleGridPos, 0.0f, (float)(gridResolution - 1));

	//iterate through neighbors of current grid cell
	glm::vec3 v1 = glm::vec3(0.0f);
	glm::vec3 v2 = glm::vec3(0.0f);
	glm::vec3 v3 = glm::vec3(0.0f);
	int neighborCount1 = 0;
	int neighborCount2 = 0;

	glm::vec3 currParticlePos = pos[index];
	glm::vec3 currParticleVel = vel1[index];

	for (int i = -1; i < 1; i++) {
		for (int j = -1; j < 1; j++) {
			for (int k = -1; k < 1; k++) {

				//[QUESTION] Do I need to clamp?
				//separate these out and imax(x + i, 0) and imin(_x, gridResolution - 1)
				//int _x = imin(imax(particleGridPos.x + i, 0), gridResolution - 1);
				//int _y = imin(imax(particleGridPos.y + j, 0), gridResolution - 1);
				//int _z = imin(imax(particleGridPos.z + k, 0), gridResolution - 1);
				//int particleGridIdx = gridIndex3Dto1D(_x, _y, _z, gridResolution);
				int particleGridIdx = gridIndex3Dto1D(particleGridPos.x + i, particleGridPos.y + j, particleGridPos.z + k, gridResolution);

				int startIdx = gridCellStartIndices[particleGridIdx];
				int endIdx = gridCellEndIndices[particleGridIdx];

				if (startIdx != -1)	//shouldn't need to check endIdx b/c startIdx and endIdx should be -1 in the same places
				{
					for (int n = startIdx; n <= endIdx; n++)
					{
						//distance between current boid in for loop and this boid that's at index
						glm::vec3 currNeighboringParticlePos = pos[n];
						glm::vec3 currNeighboringParticleVel = vel1[n];

						float distance = glm::distance(currNeighboringParticlePos, currParticlePos);

						if (n != index)
						{
							if (distance < rule1Distance)
							{
								v1 += currNeighboringParticlePos;
								neighborCount1++;
							}

							if (distance < rule2Distance)
							{
								v2 -= (currNeighboringParticlePos - currParticlePos);
							}

							if (distance < rule3Distance)
							{
								v3 += currNeighboringParticleVel;
								neighborCount2++;
							}
						}//end if n!= index
					}//end for n
				}//end if startIdx != -1
			}//end for k
		}//end for j
	}//end for i


	if (neighborCount1 > 0)
	{
		v1 /= neighborCount1;
		v1 = (v1 - currParticlePos) * rule1Scale;
	}

	if (neighborCount2 > 0)
	{
		v3 /= neighborCount2;
		v3 *= rule3Scale;
	}

	v2 *= rule2Scale;

	glm::vec3 totalVel = currParticleVel + v1 + v2 + v3;

	//Clamp the speed change
	if (glm::length(totalVel) > maxSpeed)
	{
		totalVel = glm::normalize(totalVel) * maxSpeed;
	}

	vel2[index] = totalVel;
}

// ===========================================================================================
/*
								STEP SIMULATION FUNCTIONS
*/
// ===========================================================================================


/*
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	std::swap(dev_vel1, dev_vel2);
}

// --------------------------------------------------------------------------------------------

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed


	dim3 fullBlocksPerGridParticle((numObjects + blockSize - 1) / blockSize);

	//Make sure to initialize start and end arrays to -1, and then update them in kernIdentifyCellStartEnd
	//-1 signifies no particles in that current gridcell
	//Need this one for resetintbuffers, because start and end index arrays are based on grid cell count, not num particles
	dim3 fullBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);

	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for gridCellStartIndices failed!");

	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for gridCellEndIndices failed!");

	//Calculate the particleArrayIndices and particleGridIndices
	kernComputeIndices<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, 
																	dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	hipDeviceSynchronize();

	//Sort particleArrayIndices by gridIndices
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	checkCUDAErrorWithLine("sorting by thrust key values failed!");

	//Calculate the startIdxArray and endIdxArray
	kernIdentifyCellStartEnd<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	
	//Calculate the total velocity based on neighbors
	kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, 
																					dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, 
																					dev_pos, dev_vel1, dev_vel2);

	kernUpdatePos<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	std::swap(dev_vel1, dev_vel2);
	
}

// --------------------------------------------------------------------------------------------

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.


	dim3 fullBlocksPerGridParticle((numObjects + blockSize - 1) / blockSize);

	//Make sure to initialize start and end arrays to -1, and then update them in kernIdentifyCellStartEnd
	//-1 signifies no particles in that current gridcell
	//Need this one for resetintbuffers, because start and end index arrays are based on grid cell count, not num particles
	dim3 fullBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);

	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for gridCellStartIndices failed!");

	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for gridCellEndIndices failed!");

	//Calculate the particleArrayIndices and particleGridIndices
	kernComputeIndices<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, 
		gridSideCount, gridMinimum, gridInverseCellWidth,
		dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	
	hipDeviceSynchronize();
	
	//Sort particleArrayIndices by gridIndices
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	checkCUDAErrorWithLine("sorting by thrust key values failed!");

	//Calculate the startIdxArray and endIdxArray
	kernIdentifyCellStartEnd<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, 
		dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	kernComputeCoherentBuffers<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, 
		dev_particleArrayIndices, dev_pos, dev_vel1, coherent_pos, coherent_vel);

	//Calculate the total velocity based on neighbors
	kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGridParticle, blockSize>>>(numObjects,
		gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		coherent_pos, coherent_vel, dev_vel2);

	kernUpdatePos<<<fullBlocksPerGridParticle, blockSize>>>(numObjects, dt, coherent_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	//THE SWAPS WILL BE DIFFERENT
	std::swap(coherent_pos, dev_pos);
	std::swap(coherent_vel, dev_vel2);

}

// --------------------------------------------------------------------------------------------

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.

  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(coherent_pos);
  hipFree(coherent_vel);
}

// --------------------------------------------------------------------------------------------

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);


  /*
	NOTE:
		thrust::device_ptr stores a pointer to an object allocated in device memory. 
		This type provides type safety when dispatching standard algorithms on ranges resident in device memory.
		Basically it wraps up the raw pointers for those thrust algorithm (such as sort) to use.
  */

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
