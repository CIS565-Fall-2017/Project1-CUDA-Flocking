#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// Useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// Parameters for the boids algorithm.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

//Size of the starting area in simulation space. --> -scene_scale to scene_scale in every dimension
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// These buffers are here to hold all your boid information.
// These are allocated in Boids::initSimulation.
// We need two velocity buffers in a simulation so we can ping-pong the buffers.
// This way we can modify the data of one of the velocity buffers while reading from the other
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // Stores which index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // Stores which grid cell is this particle in?

							  // Needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // Stores Which part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this grid cell?

							   // Additional buffers needed to reshuffle
							   // the position and velocity data to be coherent within cells.
glm::vec3 *dev_coherentVel; //rearranged form of dev_vel2 so that it is more memory coherent
glm::vec3 *dev_coherentPos; //rearranged form of dev_pos so that it is more memory coherent

							// Grid parameters based on simulation parameters.
							// These are computed in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/*
* Helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index)
{
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* This is a basic CUDA kernel.
* CUDA kernel for generating boids with a random position somewhere inside the simulation space
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		glm::vec3 rand = generateRandomVec3(time, index);
		arr[index].x = scale * rand.x;
		arr[index].y = scale * rand.y;
		arr[index].z = scale * rand.z;
	}
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N)
{
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize); //To ensure if N is not an exact multiple of blocksize, 
															 //the remainder of N/blocksize is still a portion of N which 
															 //would be ignored if we dont have an extra block to 
															 //accommodate the remainder of the N objects

															 // Basic CUDA memory management and error checking.
															 // Don't forget to hipFree in  Boids::endSimulation.
															 // Allocating all buffers at once is more efficient.
	hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

	//generate random initial positions for boids
	kernGenerateRandomPosArray << <fullBlocksPerGrid, blockSize >> >(1, numObjects, dev_pos, scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	//Computing grid parameters
	gridCellWidth = 0.5f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1; //not sure why + 1
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_coherentPos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_coherentPos failed!");

	hipMalloc((void**)&dev_coherentVel, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_coherentVel failed!");

	hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/*
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/*
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities)
{
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);
	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* Helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity of the boid with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel)
{
	glm::vec3 v1 = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 v2 = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 v3 = glm::vec3(0.0f, 0.0f, 0.0f);

	glm::vec3 percieved_center_of_mass = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceived_velocity = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separate_vector = glm::vec3(0.0f, 0.0f, 0.0f);

	int neighborCount1 = 0;
	int neighborCount3 = 0;

	float distance = 0.0f;

	for (int i = 0; i < N; i++)
	{
		if (i != iSelf)
		{
			// 3 rules for the basic boids algorithm
			distance = glm::distance(pos[i], pos[iSelf]);
			if (distance < rule1Distance)
			{
				percieved_center_of_mass += pos[i];
				neighborCount1++;
			}

			if (distance < rule2Distance)
			{
				separate_vector -= (pos[i] - pos[iSelf]);
			}

			if (distance < rule3Distance)
			{
				perceived_velocity += vel[i];
				neighborCount3++;
			}
		}
	}

	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	if (neighborCount1 != 0)
	{
		percieved_center_of_mass /= neighborCount1;
		v1 = (percieved_center_of_mass - pos[iSelf])*rule1Scale;
	}

	// Rule 2: boids try to stay a distance d away from each other
	v2 = separate_vector*rule2Scale;

	// Rule 3: boids try to match the speed of surrounding boids
	if (neighborCount3 != 0)
	{
		perceived_velocity /= neighborCount3;
		v3 = perceived_velocity*rule3Scale;
	}

	return v1 + v2 + v3;
}

/*
* Implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	// Compute a new velocity based on pos and vel1
	glm::vec3 newVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);
	// Clamp the speed
	if (glm::length(newVel) > maxSpeed)
	{
		newVel = glm::normalize(newVel) * maxSpeed;
	}
	// Record the new velocity into vel2. 
	// Question: why NOT vel1?
	// Answer: vel1 is being read from as well in this kernel. And so if we wrote into it some threads might read 
	// in the incorrect data or they simply may not be able to read it because it is being written into, etc. This
	// is why we ping-pog the velocity buffers
	vel2[index] = newVel;
}

/*
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel)
{
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

	thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// Method of computing a 1D index from a 3D grid index.
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution)
{
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	glm::vec3 gridMin, float inverseCellWidth,
	glm::vec3 *pos, int *indices, int *gridIndices)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	// Go through the boids and determine which grid cell to bin them into
	glm::ivec3 boidPos = (pos[index] - gridMin) * inverseCellWidth;
	gridIndices[index] = gridIndex3Dto1D(boidPos.x, boidPos.y, boidPos.z, gridResolution);
	// Set up a parallel array of integer indices as pointers to the actual
	// boid data in pos and vel1/vel2
	indices[index] = index;
}

// This is useful for indicating that a cell does not enclose any boids
// Called at the beginning of every step of a simulation to reset the buffer values to a default value which 
// tells us if the cell holds any boids or not
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N)
	{
		intBuffer[index] = value;
	}
}

// Identify the start and end points of each gridcell in the gridIndices array.
__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
	int *gridCellStartIndices, int *gridCellEndIndices)
{
	//go through particleGridIndices identifying when there is a change in there value, 
	//which signifies a change in the gridcell we are dealing with
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	if (index == 0) //edge case
	{
		gridCellStartIndices[particleGridIndices[index]] = 0;
	}
	else if (index == N - 1) //edge case
	{
		gridCellEndIndices[particleGridIndices[index]] = N - 1;
	}
	else if (particleGridIndices[index] != particleGridIndices[index + 1])
	{
		//inbetween grid cells with no boids are set to -1  --> done before when both the arrays were reset to -1

		//change in gridcell
		gridCellEndIndices[particleGridIndices[index]] = index;
		gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;
	}
}

// Store the reshuffled position and velocity buffers that are more memory coherent in new coherentPos and coherentVel buffers 
__global__ void kernSetCoherentPosVel(int N, int *particleArrayIndices,
	int *gridCellStartIndices, int *gridCellEndIndices,
	const glm::vec3 *pos, const glm::vec3 *vel,
	glm::vec3 *coherentPos, glm::vec3 *coherentVel)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	int coherentindex = particleArrayIndices[index];

	coherentPos[index] = pos[coherentindex];
	coherentVel[index] = vel[coherentindex];
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	int *particleArrayIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2)
{
	// Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	//find boid position
	//then use that position to determine the grid cell the boid belongs to
	//use that information to find the 8 cells you have to check
	glm::ivec3 boidPos = (pos[index] - gridMin) * inverseCellWidth;
	int x = boidPos.x;
	int y = boidPos.y;
	int z = boidPos.z;

	glm::vec3 v1 = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 v2 = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 v3 = glm::vec3(0.0f, 0.0f, 0.0f);

	glm::vec3 percieved_center_of_mass = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceived_velocity = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separate_vector = glm::vec3(0.0f, 0.0f, 0.0f);

	int neighborCount1 = 0;
	int neighborCount3 = 0;

	float distance = 0.0f;

	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			for (int k = -1; k <= 1; k++)
			{
				int _x = x + i;
				int _y = y + j;
				int _z = z + k;

				_x = imax(_x, 0);
				_y = imax(_y, 0);
				_z = imax(_z, 0);

				_x = imin(_x, gridResolution - 1);
				_y = imin(_y, gridResolution - 1);
				_z = imin(_z, gridResolution - 1);

				int boidGridCellindex = gridIndex3Dto1D(_x, _y, _z, gridResolution);

				// Identify which cells may contain neighboring boids. This isn't always 8.
				// SWITCHED to identifying upto 27 neighboring cells containing boids 
				if (gridCellStartIndices[boidGridCellindex] != -1)
				{
					//we know the grid cell is empty if its start or end indices have been set to -1

					// For each cell that contains boids and needs to be checked, 
					// read the start/end indices in the boid pointer array.

					// Now go through the boids in that grid cell and apply the rules 
					// to it if it falls within the neighbor hood distance
					for (int h = gridCellStartIndices[boidGridCellindex]; h <= gridCellEndIndices[boidGridCellindex]; h++)
					{
						//Access each boid in the cell and compute velocity change from
						int bindex = particleArrayIndices[h];
						if (h != index)
						{
							//Compute velocity change based on rules
							distance = glm::distance(pos[bindex], pos[index]);
							if (distance < rule1Distance)
							{
								percieved_center_of_mass += pos[bindex];
								neighborCount1++;
							}

							if (distance < rule2Distance)
							{
								separate_vector -= (pos[bindex] - pos[index]);
							}

							if (distance < rule3Distance)
							{
								perceived_velocity += vel1[bindex];
								neighborCount3++;
							}
						}
					}
				}
			}
		}
	}

	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	if (neighborCount1 != 0)
	{
		percieved_center_of_mass /= neighborCount1;
		v1 = (percieved_center_of_mass - pos[index])*rule1Scale;
	}

	// Rule 2: boids try to stay a distance d away from each other
	v2 = separate_vector*rule2Scale;

	// Rule 3: boids try to match the speed of surrounding boids
	if (neighborCount3 != 0)
	{
		perceived_velocity /= neighborCount3;
		v3 = perceived_velocity*rule3Scale;
	}

	glm::vec3 newVel = vel1[index] + v1 + v2 + v3;

	// Clamp the speed change before putting the new speed in vel2
	if (glm::length(newVel) > maxSpeed)
	{
		newVel = glm::normalize(newVel) * maxSpeed;
	}
	vel2[index] = newVel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	glm::vec3 *coherentPos, glm::vec3 *coherentVel, glm::vec3 *vel2)
{
	// Very similar to kernUpdateVelNeighborSearchScattered, except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer directly to coherentPos and CoherentVel.	

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	//find boid position
	//then use that position to determine the gridcell it belongs to
	//use that information to find the 8 cells you have to check
	glm::ivec3 boidPos = (coherentPos[index] - gridMin) * inverseCellWidth;
	int x = boidPos.x;
	int y = boidPos.y;
	int z = boidPos.z;

	glm::vec3 v1 = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 v2 = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 v3 = glm::vec3(0.0f, 0.0f, 0.0f);

	glm::vec3 percieved_center_of_mass = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 perceived_velocity = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separate_vector = glm::vec3(0.0f, 0.0f, 0.0f);

	int neighborCount1 = 0;
	int neighborCount3 = 0;

	float distance = 0.0f;

	// DIFFERENCE: For best results, consider what order the cells should be
	// checked in to maximize the memory benefits of reordering the boids data.
	for (int k = -1; k <= 1; k++) //z axis
	{
		for (int j = -1; j <= 1; j++) //y axis
		{
			for (int i = -1; i <= 1; i++) //x axis
			{
				int _x = x + i;
				int _y = y + j;
				int _z = z + k;

				_x = imax(_x, 0);
				_y = imax(_y, 0);
				_z = imax(_z, 0);

				_x = imin(_x, gridResolution - 1);
				_y = imin(_y, gridResolution - 1);
				_z = imin(_z, gridResolution - 1);

				int boidGridCellindex = gridIndex3Dto1D(_x, _y, _z, gridResolution);

				// Identify which cells may contain neighbors. This isn't always 8.
				// For each cell, read the start/end indices in the boid pointer array.

				if (gridCellStartIndices[boidGridCellindex] != -1)
				{
					//we know the grid cell is empty if its start or end indices have been set to -1

					//now go through the boids in that grid cell and apply the rules 
					//to it if it falls within the neighbor hood distance
					for (int h = gridCellStartIndices[boidGridCellindex]; h <= gridCellEndIndices[boidGridCellindex]; h++)
					{
						if (h != index)
						{
							// Access each boid in the cell and compute velocity change from
							// the boids rules, if this boid is within the neighborhood distance.
							distance = glm::distance(coherentPos[h], coherentPos[index]);
							if (distance < rule1Distance)
							{
								percieved_center_of_mass += coherentPos[h];
								neighborCount1++;
							}

							if (distance < rule2Distance)
							{
								separate_vector -= (coherentPos[h] - coherentPos[index]);
							}

							if (distance < rule3Distance)
							{
								perceived_velocity += coherentVel[h];
								neighborCount3++;
							}
						}
					}
				}
			}
		}
	}

	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	if (neighborCount1 != 0)
	{
		percieved_center_of_mass /= neighborCount1;
		v1 = (percieved_center_of_mass - coherentPos[index])*rule1Scale;
	}

	// Rule 2: boids try to stay a distance d away from each other
	v2 = separate_vector*rule2Scale;

	// Rule 3: boids try to match the speed of surrounding boids
	if (neighborCount3 != 0)
	{
		perceived_velocity /= neighborCount3;
		v3 = perceived_velocity*rule3Scale;
	}

	glm::vec3 newVel = coherentVel[index] + v1 + v2 + v3;

	// Clamp the speed change before putting the new speed in vel2
	if (glm::length(newVel) > maxSpeed)
	{
		newVel = glm::normalize(newVel) * maxSpeed;
	}
	vel2[index] = newVel;
}


//Step the entire N-body simulation by `dt` seconds.
void Boids::stepSimulationNaive(float dt)
{
	//Step the simulation forward in time.
	//Setup thread/block execution configuration
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);//no dim1,  dim3 automatically makes the ther dimensions 0

																	 //update boid velocities
	kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

	//update boid positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel1);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	//Ping-pong the velocity buffers
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt)
{
	// Uniform Grid Neighbor search using Thrust sort.

	dim3 fullBlocksPerGrid_gridsize((gridCellCount + blockSize - 1) / blockSize);//no dim1,  dim3 automatically makes the ther dimensions 0
	dim3 fullBlocksPerGrid_boids((numObjects + blockSize - 1) / blockSize);

	// Reset buffers start and end indices buffers
	kernResetIntBuffer << <fullBlocksPerGrid_gridsize, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGrid_gridsize, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

	// Label each particle with its array index as well as its grid index.
	// Use 2x width grids.
	// recompute grid cell indices and particlearray indices every timestep
	kernComputeIndices << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, gridSideCount,
		gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// Now sort the dev_particleGridIndices so that boids belonging to the same grid cell
	// are next to each other in the gridIndices array --> Use Thrust to sort the array

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	// Unstable key sort using Thrust. A stable sort isn't necessary
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	checkCUDAErrorWithLine("thrust sorting failed!");

	// assuming the boidGridIndices are sorted, assign values to the arrays keeping 
	// track of the data in dev_particleArrayIndices for each cell.

	// unroll the loop for finding the start and end indices of each
	// cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, dev_particleGridIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices);

	// Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

	// Update positions
	kernUpdatePos << <fullBlocksPerGrid_boids, blockSize >> >(numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	//Ping-pong the velocity buffers
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt)
{
	// Uniform Grid Neighbor search using Thrust sort with cell-coherent data.

	dim3 fullBlocksPerGrid_gridsize((gridCellCount + blockSize - 1) / blockSize);//no dim1,  dim3 automatically makes the ther dimensions 0
	dim3 fullBlocksPerGrid_boids((numObjects + blockSize - 1) / blockSize);

	//Reset buffers start and end indices buffers
	kernResetIntBuffer << <fullBlocksPerGrid_gridsize, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGrid_gridsize, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
	// Label each particle with its array index as well as its grid index.
	// Use 2x width grids
	// recompute grid cell indices and particlearray indices every timestep
	kernComputeIndices << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, gridSideCount,
		gridMinimum, gridInverseCellWidth,
		dev_pos, dev_particleArrayIndices,
		dev_particleGridIndices);
	// Now sort the dev_particleGridIndices so that boids belonging to the same grid cell
	// are next to each other in the gridIndices array --> Use Thrust to sort the array
	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	// Unstable key sort using Thrust. A stable sort isn't necessary
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	checkCUDAErrorWithLine("thrust sorting failed!");
	// Assuming the boidGridIndices are sorted, assign values to the arrays keeping 
	// track of the data in dev_particleArrayIndices for each cell.
	// Unroll the loop for finding the start and end indices of each
	// cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, dev_particleGridIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices);
	// BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all the boid data(position and velocity)
	// in the simulation array, such that it is memory coherent arranged in order of grid cells
	kernSetCoherentPosVel << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, dev_particleArrayIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_pos, dev_vel1, dev_coherentPos, dev_coherentVel);
	// Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid_boids, blockSize >> > (numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_coherentPos, dev_coherentVel, dev_vel2);

	// Update positions
	kernUpdatePos << <fullBlocksPerGrid_boids, blockSize >> >(numObjects, dt, dev_coherentPos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");
	// Ping-pong Coherent and regular pos buffers
	std::swap(dev_coherentPos, dev_pos);
	// Ping-pong the velocity buffers
	std::swap(dev_vel1, dev_vel2);
}
//Free memory that was allocated in initSimulation
void Boids::endSimulation()
{
	//Free any buffers here
	hipFree(dev_vel1);
	hipFree(dev_vel2);
	hipFree(dev_pos);
	hipFree(dev_particleArrayIndices);
	hipFree(dev_particleGridIndices);
	hipFree(dev_gridCellStartIndices);
	hipFree(dev_gridCellEndIndices);
	hipFree(dev_coherentPos);
	hipFree(dev_coherentVel);
}

void Boids::unitTest()
{
	// Test unstable sort
	int *dev_intKeys;
	int *dev_intValues;
	int N = 10;
	int *intKeys = new int[N];
	int *intValues = new int[N];
	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;
	hipMalloc((void**)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");
	hipMalloc((void**)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++)
	{
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}
	// Copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);
	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);
	// Copy data back to the CPU side from the GPU
	hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");
	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++)
	{
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}
	// Cleanup
	delete[] intKeys;
	delete[] intValues;
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}