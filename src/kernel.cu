#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_pos_rearranged;
glm::vec3 *dev_vel_rearranged;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices  failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices  failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices  failed!");

  hipMalloc((void**)&dev_pos_rearranged, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_rearranged failed!");

  hipMalloc((void**)&dev_vel_rearranged, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel_rearranged failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
	glm::vec3 v1(0);
	glm::vec3 v2(0); 
	glm::vec3 v3(0);

	int num1 = 0;
	int num2 = 0;
	
	for (int i = 0; i < N; i++) {
		//float distance = (pos[i].x - pos[iSelf].x) * (pos[i].x - pos[iSelf].x) + (pos[i].y - pos[iSelf].y) * (pos[i].y - pos[iSelf].y) + (pos[i].z - pos[iSelf].z) * (pos[i].z - pos[iSelf].z);
		float distance = glm::length(pos[i] - pos[iSelf]);
		if (i != iSelf) {
			// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
			if (distance < rule1Distance) {
				v1 += pos[i];
				num1++;
			}
			// Rule 2: boids try to stay a distance d away from each other
			if(distance < rule2Distance)
				v2 -= (pos[i] - pos[iSelf]);
			// Rule 3: boids try to match the speed of surrounding boids
			if (distance < rule2Distance) {
				v3 += vel[i];
				num2++;
			}
			
		}
	}
	if(num1)
		v1 = (v1 / float(num1) - pos[iSelf])* rule1Scale;
	v2 = v2 * rule2Scale;
	if(num2)
		v3 = (v3 / float(num2) - vel[iSelf])* rule3Scale;

  return v1 + v2 + v3;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
		return;
	glm::vec3 vel_change;
	vel_change = computeVelocityChange(N, index, pos, vel1);
	vel2[index] = vel1[index] + vel_change;
	vel2[index] = glm::length(vel2[index]) > maxSpeed ? glm::normalize(vel2[index]) * maxSpeed : vel2[index];

  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	int x, y, z;
	x = (pos[index].x - gridMin.x) * inverseCellWidth;
	y = (pos[index].y - gridMin.y) * inverseCellWidth;
	z = (pos[index].z - gridMin.z) * inverseCellWidth;
	gridIndices[index] = gridIndex3Dto1D(x, y, z, gridResolution);
	indices[index] = index;
	// - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	int l = -1;
	if (index >= N)
		return;
	if (index == 0)
	{
		gridCellStartIndices[particleGridIndices[0]] = 0;
	}
	if (index == N - 1)
		gridCellEndIndices[particleGridIndices[N - 1]] = N - 1;
	else if (particleGridIndices[index] != particleGridIndices[index + 1])
	{
		gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;
		gridCellEndIndices[particleGridIndices[index]] = index;
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	int boid_Index = particleArrayIndices[index];

	float x, y, z;
	x = (pos[boid_Index].x - gridMin.x)*inverseCellWidth;
	y = (pos[boid_Index].y - gridMin.y)*inverseCellWidth;
	z = (pos[boid_Index].z - gridMin.z)*inverseCellWidth;
	
	int grid_Index = gridIndex3Dto1D((int)x, (int)y, (int)z, gridResolution);

	int flagx = int(x + 0.5f) - int(x) ? 0 : -1;
	int flagy = int(y + 0.5f) - int(y) ? 0 : -1;
	int flagz = int(z + 0.5f) - int(z) ? 0 : -1;
	int GridCellNum = gridResolution * gridResolution * gridResolution;
	
	glm::vec3 v1(0), v2(0), v3(0);

	int num1 = 0;
	int num2 = 0;
	for(int i = flagx; i <= flagx + 1; i++)
		for(int j = flagy; j <= flagy + 1; j++)
			for (int k = flagz; k <= flagz + 1; k++) {
				int g_idx;
				g_idx = grid_Index + i + j * gridResolution + k * gridResolution * gridResolution;
				int startIndex = gridCellStartIndices[g_idx];
				int endIndex = gridCellEndIndices[g_idx];
				if (startIndex == -1 && endIndex == -1 && g_idx >= 0 && g_idx < GridCellNum)
					continue;
				for (int m = startIndex; m <= endIndex; m++) {
					int idx = particleArrayIndices[m];
					if (idx != boid_Index) {
						float distance = glm::length(pos[idx] - pos[boid_Index]);
						// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
						if (distance < rule1Distance) {
							v1 += pos[idx];
							num1++;
						}
						// Rule 2: boids try to stay a distance d away from each other
						if (distance < rule2Distance)
							v2 -= (pos[idx] - pos[boid_Index]);
						// Rule 3: boids try to match the speed of surrounding boids
						if (distance < rule3Distance) {
							v3 += vel1[idx];
							num2++;
						}
					}
				}
			}
	if (num1)
		v1 = (v1 / float(num1) - pos[boid_Index]) * rule1Scale;
	v2 = v2 * rule2Scale;
	if (num2)
		v3 = (v3 / float(num2) - vel1[boid_Index])* rule3Scale;
	glm::vec3 v_change = v1 + v2 + v3;
	
	vel2[boid_Index] = vel1[boid_Index] + v_change;// + v2 + v3;

	vel2[boid_Index] = glm::length(vel2[boid_Index]) > maxSpeed ? glm::normalize(vel2[boid_Index]) * maxSpeed : vel2[boid_Index];
}

__global__ void kernRearrangePosVel(int N, int* particleArrayIndices, glm::vec3* pos_rearranged, glm::vec3* pos, glm::vec3* vel_rearranged, glm::vec3* vel){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	int idx = particleArrayIndices[index];
	pos_rearranged[index] = pos[idx];
	vel_rearranged[index] = vel[idx];
}
__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	//int boid_Index = particleArrayIndices[index];
	glm::vec3 thispos = pos[index];

	float x, y, z;
	x = (thispos.x - gridMin.x)*inverseCellWidth;
	y = (thispos.y - gridMin.y)*inverseCellWidth;
	z = (thispos.z - gridMin.z)*inverseCellWidth;

	int grid_Index = gridIndex3Dto1D((int)x, (int)y, (int)z, gridResolution);

	int flagx = int(x + 0.5f) - int(x) ? 0 : -1;
	int flagy = int(y + 0.5f) - int(y) ? 0 : -1;
	int flagz = int(z + 0.5f) - int(z) ? 0 : -1;
	int GridCellNum = gridResolution * gridResolution * gridResolution;

	glm::vec3 v1(0), v2(0), v3(0);

	int num1 = 0;
	int num2 = 0;
	for (int i = flagx; i <= flagx + 1; i++)
		for (int j = flagy; j <= flagy + 1; j++)
			for (int k = flagz; k <= flagz + 1; k++) {
				int g_idx;
				g_idx = grid_Index + i + j * gridResolution + k * gridResolution * gridResolution;
				int startIndex = gridCellStartIndices[g_idx];
				int endIndex = gridCellEndIndices[g_idx];
				if (startIndex == -1 && endIndex == -1 && g_idx >= 0 && g_idx < GridCellNum)
					continue;
				for (int m = startIndex; m <= endIndex; m++) {
					if (m != index) {
						float distance = glm::length(pos[m] - thispos);
						// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
						if (distance < rule1Distance) {
							v1 += pos[m];
							num1++;
						}
						// Rule 2: boids try to stay a distance d away from each other
						if (distance < rule2Distance)
							v2 -= (pos[m] - thispos);
						// Rule 3: boids try to match the speed of surrounding boids
						if (distance < rule3Distance) {
							v3 += vel1[m];
							num2++;
						}
					}
				}
			}
	if (num1)
		v1 = (v1 / float(num1) - pos[index]) * rule1Scale;
	v2 = v2 * rule2Scale;
	if (num2)
		v3 = (v3 / float(num2) - vel1[index])* rule3Scale;
	glm::vec3 v_change = v1 + v2 + v3;

	vel2[index] = vel1[index] + v_change;// + v2 + v3;

	vel2[index] = glm::length(vel2[index]) > maxSpeed ? glm::normalize(vel2[index]) * maxSpeed : vel2[index];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	dim3 blocks((numObjects + blockSize - 1) / blockSize);
	
	kernUpdateVelocityBruteForce << <blocks, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
	std::swap(dev_vel1, dev_vel2);
	kernUpdatePos << <blocks, blockSize >> > (numObjects, dt, dev_pos, dev_vel1);
	// TODO-1.2 ping-pong the velocity buffers
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
//Compute Indices
	dim3 blocks((numObjects + blockSize - 1) / blockSize);
	kernComputeIndices<< <blocks, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
//Sort
	//keys
	thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices);
	//values
	thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices);

	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
//Compute Start End indices
	dim3 gridBlocks((gridCellCount + blockSize - 1) / blockSize);
	kernResetIntBuffer << <gridBlocks, blockSize >> >(gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <gridBlocks, blockSize >> >(gridCellCount, dev_gridCellEndIndices, -1);

	kernIdentifyCellStartEnd<<<blocks, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

//Update velocity(Func)
	kernUpdateVelNeighborSearchScattered<<<blocks, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
//ping-pong buffers
	std::swap(dev_vel1, dev_vel2);
//Update position
	kernUpdatePos << <blocks, blockSize >> >(numObjects, dt, dev_pos, dev_vel1);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
  //Compute Indices
	dim3 blocks((numObjects + blockSize - 1) / blockSize);
	kernComputeIndices << <blocks, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	//Sort
	//keys
	thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices);
	//values
	thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices);

	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
	
	
	//Compute Start End indices
	dim3 gridBlocks((gridCellCount + blockSize - 1) / blockSize);
	kernResetIntBuffer<<<gridBlocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer<<<gridBlocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

	kernIdentifyCellStartEnd << <blocks, blockSize >> >(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	//Rearrange
	kernRearrangePosVel << <blocks, blockSize >> > (numObjects, dev_particleArrayIndices, dev_pos_rearranged, dev_pos, dev_vel_rearranged, dev_vel1);

	//Update velocity(Func)
	kernUpdateVelNeighborSearchCoherent << <blocks, blockSize >> >(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, 
		dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos_rearranged, dev_vel_rearranged, dev_vel1);
	
	std::swap(dev_pos, dev_pos_rearranged);

	//Update position
	kernUpdatePos << <blocks, blockSize >> >(numObjects, dt, dev_pos, dev_vel1);
	//ping-pong buffers
	//hipMemcpy(dev_pos, dev_pos_rearranged, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_pos_rearranged);
  hipFree(dev_vel_rearranged);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete[] intKeys;
  delete[] intValues;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
